#include "hip/hip_runtime.h"
#include "fd.h"
#include "time.h"
#include "misc.h"

extern "C" {
	#include "cwp.h"
}
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#define sizeblock 8 

void cudaCheck(){
    hipError_t err=hipGetLastError();
    if(err!=hipSuccess){
      printf("%s\n", hipGetErrorString(err));
      exit(1);
    }
}


#ifdef PERF_COUNTERS
	int wrTransferCnt;
	int rdTransferCnt;
	float fwAVGTime;
	float bwAVGTime;
	// float fwDeviceAVGTime;
	// float bwDeviceAVGTime;
	float kernelAVGTime;
	float wrAVGTime;
	float rdAVGTime;
	float deviceAVGTime; // rd+wr+kernel

void fd_print_report(int nx, int nz) {
printf("> ================================================ \n");
	printf("> Exec Time Report (NX = %d NZ= %d):\n",nx,nz);
	printf("> Device       = %.1f (s)\n",deviceAVGTime/1000000.0);
	// printf("> Total        = %.1f (s)\n",execTime/1000000.0);
	// printf("> WR TransfCnt = %d \n",wrTransferCnt/1000000.0);
	// printf("> Write        = %.1f (%.2f%%)(s)\n",wrAVGTime/1000000.0, 
	// 	(100.0*wrAVGTime/deviceAVGTime));
	// printf("> RD TransfCnt = %d \n",rdTransferCnt/1000000.0);
	// printf("> Read         = %.1f (%.2f%%)(s)\n",rdAVGTime/1000000.0,
	// 	(100.0*rdAVGTime/deviceAVGTime));
	printf("> Device Fwrd  = %.1f (s)\n",fwAVGTime/1000000.0);
	printf("> Device Bwrd  = %.1f (s)\n",bwAVGTime/1000000.0);
}
#endif
//////////////////////////////////////////
// Cuda enviroroment global variables  
float *d_p, *d_pr, *d_pp, *d_ppr, *d_swap; 
float *d_laplace, *d_v2, *d_coefs_x, *d_coefs_z;
float *d_taperx, *d_taperz, *d_upb, *d_sis, *d_img;

size_t mtxBufferLength, brdBufferLength;
size_t imgBufferLength, obsBufferLength; 
size_t coefsBufferLength, upbBufferLength;

float *taper_x, *taper_z; 
int nxbin, nzbin; 

int gridx, gridz, gridupb;
int gridBorder_x, gridBorder_z;

static float dx2inv,dz2inv,dt2;
static float **laplace = NULL;
static float *coefs = NULL;
static float *coefs_z = NULL;
static float *coefs_x = NULL;
static void makeo2 (float *coef,int order);

float *calc_coefs(int order);

void fd_init_cuda(int order, int nxe, int nze, 
	int nxb, int nzb, int nt, int ns, float fac){
	float dfrac; 
	// cudaProfilerSart();
   	nxbin=nxb; nzbin=nzb; 
   	brdBufferLength = nxb*sizeof(float);
   	mtxBufferLength = (nxe*nze)*sizeof(float);
   	coefsBufferLength = (order+1)*sizeof(float);
   	upbBufferLength = nt*nxe*(order/2)*sizeof(float);
	obsBufferLength = nt*(nxe-(2*nxb))*sizeof(float);
   	imgBufferLength = (nxe-(2*nxb))*(nze-(2*nzb))*sizeof(float);

	taper_x = alloc1float(nxb);
	taper_z = alloc1float(nzb);

	dfrac = sqrt(-log(fac))/(1.*nxb);
	for(int i=0;i<nxb;i++)
	  taper_x[i] = exp(-pow((dfrac*(nxb-i)),2));


	dfrac = sqrt(-log(fac))/(1.*nzb);
	for(int i=0;i<nzb;i++)
	  taper_z[i] = exp(-pow((dfrac*(nzb-i)),2));

    
	// Create a Device pointers
	hipMalloc((void **) &d_v2, mtxBufferLength);
	hipMalloc((void **) &d_p, mtxBufferLength);
	hipMalloc((void **) &d_pp, mtxBufferLength);
	hipMalloc((void **) &d_pr, mtxBufferLength);
	hipMalloc((void **) &d_ppr, mtxBufferLength);
	hipMalloc((void **) &d_swap, mtxBufferLength);
	hipMalloc((void **) &d_laplace, mtxBufferLength);
	
	hipMalloc((void **) &d_upb, upbBufferLength);
	hipMalloc((void **) &d_sis, obsBufferLength);
	hipMalloc((void **) &d_img, imgBufferLength);
	hipMalloc((void **) &d_coefs_x, coefsBufferLength);
	hipMalloc((void **) &d_coefs_z, coefsBufferLength);
	hipMalloc((void **) &d_taperx, brdBufferLength);
	hipMalloc((void **) &d_taperz, brdBufferLength);

	int div_x, div_z; 
	// Set a Grid for the execution on the device
	div_x = (float) nxe/(float) sizeblock; 
	div_z = (float) nze/(float) sizeblock; 
	gridx = (int) ceil(div_x);
	gridz = (int) ceil(div_z);

	div_x = (float) nxb/(float) sizeblock; 
	div_z = (float) nzb/(float) sizeblock; 
	gridBorder_x = (int) ceil(div_x); 
	gridBorder_z = (int) ceil(div_z); 
	
	div_x = (float) 8/(float) sizeblock; 
	gridupb = (int) ceil(div_x); 
}

void fd_init(int order, int nx, int nz, int nxb, int nzb, 
	int nt, int ns, float fac, float dx, float dz, float dt){
	int io;
	dx2inv = (1./dx)*(1./dx);
    dz2inv = (1./dz)*(1./dz);
	dt2 = dt*dt;

	coefs = calc_coefs(order);
	laplace = alloc2float(nz,nx);

	coefs_z = calc_coefs(order);
	coefs_x = calc_coefs(order);

	// pre calc coefs 8 d2 inv
	for (io = 0; io <= order; io++) {
		coefs_z[io] = dz2inv * coefs[io];
		coefs_x[io] = dx2inv * coefs[io];
	}
	
	memset(*laplace,0,nz*nx*sizeof(float));

 	#ifdef CUDA
        fd_init_cuda(order,nx,nz,nxb,nzb,nt,ns,fac); 
    #endif

#ifdef PERF_COUNTERS
	wrTransferCnt=0;
	rdTransferCnt=0;
	kernelAVGTime=0.;
	deviceAVGTime=0.;
	wrAVGTime=0.;
	rdAVGTime=0.;
	fwAVGTime=0.;
	bwAVGTime=0.;
#endif	

	return;
}

void fd_reinit(int order, int nx, int nz){
   // todo: free coefs or realloc
	int io;
	coefs = calc_coefs(order);
	coefs_z = calc_coefs(order);
	coefs_x = calc_coefs(order);
	// pre calc coefs 8 d2inv
	for (io = 0; io <= order; io++) {
		coefs_z[io] = dz2inv * coefs[io];
		coefs_x[io] = dx2inv * coefs[io];
	}	
	// todo: free laplace or realloc
	laplace = alloc2float(nz,nx);
	memset(*laplace,0,nz*nx*sizeof(float));
	return;
}

void fd_destroy(){
	free2float(laplace);
	free1float(coefs);
	#ifdef CUDA
		hipFree(d_p);
		hipFree(d_pp);
		hipFree(d_pr);
		hipFree(d_ppr);
		hipFree(d_v2);
		hipFree(d_laplace);
		hipFree(d_coefs_z);
		hipFree(d_coefs_x);
		
		hipFree(d_taperx);
		hipFree(d_taperz);
		
		hipFree(d_sis);
		hipFree(d_img);
		hipFree(d_upb);

	#endif
	return;
}

float *calc_coefs(int order){
	float *coef;

	coef = (float *)calloc(order+1,sizeof(float));

	switch(order){
		case 2:
			coef[0] = 1.;
			coef[1] = -2.;
			coef[2] = 1.;
			break;
		case 4:
			coef[0] = -1./12.;
			coef[1] = 4./3.;
			coef[2] = -5./2.;
			coef[3] = 4./3.;
			coef[4] = -1./12.;
			break;
		case 6:
			coef[0] = 1./90.;
			coef[1] = -3./20.;
			coef[2] = 3./2.;
			coef[3] = -49./18.;
			coef[4] = 3./2.;
			coef[5] = -3./20.;
			coef[6] = 1./90.;
			break;
		case 8:
			coef[0] = -1./560.;
			coef[1] = 8./315.;
			coef[2] = -1./5.;
			coef[3] = 8./5.;
			coef[4] = -205./72.;
			coef[5] = 8./5.;
			coef[6] = -1./5.;
			coef[7] = 8./315.;
			coef[8] = -1./560.;
			break;
		default:
			makeo2(coef,order);
	}

	return coef;
}

static void makeo2 (float *coef,int order){
	float h_beta, alpha1=0.0;
	float alpha2=0.0;
	float  central_term=0.0; 
	float coef_filt=0; 
	float arg=0.0; 
	float  coef_wind=0.0;
	int msign,ix; 
	float alpha = .54;
	float beta = 6.;

	h_beta = 0.5*beta;
	alpha1=2.*alpha-1.0;
	alpha2=2.*(1.0-alpha);
	central_term=0.0;

	msign=-1;

	for (ix=1; ix <= order/2; ix++){      
		msign=-msign ;            
		coef_filt = (2.*msign)/(ix*ix); 
		arg = PI*ix/(2.*(order/2+2));
		coef_wind=pow((alpha1+alpha2*cos(arg)*cos(arg)),h_beta);
		coef[order/2+ix] = coef_filt*coef_wind;
		central_term = central_term + coef[order/2+ix]; 
		coef[order/2-ix] = coef[order/2+ix]; 
	}
	
	coef[order/2]  = -2.*central_term;

	return; 
}

__global__ void kernel_lap(int order, int nx, int nz, float * __restrict__ p, 
	float * __restrict__ lap, float * __restrict__ coefsx, float * __restrict__ coefsz){  	

   int half_order=order/2;  
  	int i =  half_order + blockIdx.x * blockDim.x + threadIdx.x; // Global row index    
  	int j =  half_order + blockIdx.y * blockDim.y + threadIdx.y; // Global column index
  	int mult = i*nz; 
  	int aux; 
	float acmx = 0, acmz = 0;
	
  	if(i<nx - half_order){
  		if(j<nz - half_order){
			for(int io=0;io<=order;io++){
				aux = io-half_order;
				acmz += p[mult + j+aux]*coefsz[io];
				acmx += p[(i+aux)*nz + j]*coefsx[io];
			}
			lap[mult +j] = acmz + acmx;
			acmx = 0.0;
			acmz = 0.0;
		}
  	}  
}

__global__ void kernel_time(int nx, int nz, float *__restrict__ p, float *__restrict__ pp,
	float *__restrict__ v2, float *__restrict__ lap, float dt2){  	

  	int i =  blockIdx.x * blockDim.x + threadIdx.x; // Global row index    
  	int j =  blockIdx.y * blockDim.y + threadIdx.y; // Global column index
  	int mult = i*nz; 
	
  	if(i<nx){
  		if(j<nz){
			 pp[mult+j] = 2.*p[mult+j] - pp[mult+j] + v2[mult+j]*dt2*lap[mult+j];		
		}
  	}  
}	

__global__ void kernel_tapper(int nx, int nz, int nxb, int nzb, 
	float *__restrict__ p, float *__restrict__ pp, float *__restrict__ taperx, float *__restrict__ taperz){  	

	int i =  blockIdx.x * blockDim.x + threadIdx.x; // nx index    
	int j =  blockIdx.y * blockDim.y + threadIdx.y; // nzb index
	int itxr = nx - 1, mult = i*nz; 

	if(i<nx){
		if(j<nzb){
			p[mult+j] *= taperz[j];
			pp[mult+j] *= taperz[j];
		}
	}

	if(i<nxb){
		if(j<nzb){
			p[mult+j] *= taperx[i];
			pp[mult+j] *= taperx[i];

			p[(itxr-i)*nz+j] *= taperx[i];
			pp[(itxr-i)*nz+j] *= taperx[i];
		}
	}
}	

__global__ void kernel_src(int nz, float * __restrict__ pp, int sx, int sz, float srce){
 	pp[sx*nz+sz] += srce;
}

__global__ void kernel_upb(int order, int nx, int nz, int nzb, int nt, float *__restrict__ pp,
	float *__restrict__ upb, int it, int flag){
	int half_order = order/2; 
	int i = blockIdx.x * blockDim.x + threadIdx.x; //nx index    
 	
 	if(i<nx){
		for(int j=nzb-order/2;j<nzb;j++)
    		if(flag == 0)
    			upb[(it*nx*half_order)+(i*half_order)+(j-(nzb-half_order))] = pp[i*nz+j];
        	else
	        	pp[i*nz+j] = upb[((nt-1-it)*nx*half_order)+(i*half_order)+(j-(nzb-half_order))];    
  	}
}

__global__ void kernel_sism(int nx, int nz, int nxb,
	int nt, int is, int it, int gz, float *__restrict__ d_obs, float *__restrict__ ppr){
 	int size = nx-(2*nxb); 
	int i = blockIdx.x * blockDim.x + threadIdx.x; //nx index    
 	if(i<size)
 		ppr[((i+nxb)*nz) + gz] += d_obs[i*nt + (nt-1-it)]; 

}

__global__ void kernel_img(int nx, int nz, int nxb, int nzb,
	float * __restrict__ imloc, float * __restrict__ p, float * __restrict__ ppr){
 	int size_x = nx-(2*nxb); 
 	int size_z = nz-(2*nzb); 
	int i =  blockIdx.x * blockDim.x + threadIdx.x; // Global row index    
  	int j =  blockIdx.y * blockDim.y + threadIdx.y; // Global column index
 	if(j<size_z){
      if(i<size_x){
        imloc[i*size_z+j] += p[(i+nxb)*nz+(j+nzb)] * ppr[(i+nxb)*nz+(j+nzb)];          
      }
    }
}

void write_buffers(float **p, float **pp, float **v2, float ***upb, 
    float *taperx, float *taperz, float **d_obs, float **imloc, int is, int flag){
	
	if(flag == 0){
		hipMemcpy(d_p, p[0], mtxBufferLength, hipMemcpyHostToDevice);
		hipMemcpy(d_pp, pp[0], mtxBufferLength, hipMemcpyHostToDevice);
		hipMemcpy(d_v2, v2[0], mtxBufferLength, hipMemcpyHostToDevice);
		hipMemcpy(d_coefs_x, coefs_x, coefsBufferLength, hipMemcpyHostToDevice);
		hipMemcpy(d_coefs_z, coefs_z, coefsBufferLength, hipMemcpyHostToDevice);
		hipMemcpy(d_taperx, taperx, brdBufferLength, hipMemcpyHostToDevice);
		hipMemcpy(d_taperz, taperz, brdBufferLength, hipMemcpyHostToDevice);
		hipMemcpy(d_upb, upb[0][0], upbBufferLength, hipMemcpyHostToDevice);
	}

	if(flag == 1){
		hipMemcpy(d_pr, p[0], mtxBufferLength, hipMemcpyHostToDevice);
		hipMemcpy(d_ppr, pp[0], mtxBufferLength, hipMemcpyHostToDevice);
		hipMemcpy(d_sis, d_obs[is], obsBufferLength, hipMemcpyHostToDevice);	
		hipMemcpy(d_img, imloc[0], imgBufferLength, hipMemcpyHostToDevice);	
	}
}

void fd_forward(int order, float **p, float **pp, float **v2, 
   float ***upb, int nz, int nx, int nt, int is, int sz, int *sx, float *srce, int propag){
	int elapsed;
    struct timeval st, et, stCR, etCR, stCW, etCW, stK, etK;
    //Start total time 
    gettimeofday(&st, NULL);

	dim3 dimGrid(gridx, gridz);	
  	dim3 dimGridTaper(gridx, gridBorder_z); 
  	
  	dim3 dimGridSingle(1,1); 
  	dim3 dimGridUpb(gridx,1); 
  	
  	dim3 dimBlock(sizeblock, sizeblock);
  	// Start write time
   	gettimeofday(&stCW, NULL);
	write_buffers(p,pp,v2,upb,taper_x, taper_z,NULL, NULL,is,0);
	// Calc avg write time  
   	gettimeofday(&etCW, NULL);
   	elapsed = ((etCW.tv_sec - stCW.tv_sec) * 1000000) + (etCW.tv_usec - stCW.tv_usec);
   	wrAVGTime += (elapsed*1.0);
   	wrTransferCnt++; 

   	// start Kernel time  
   	gettimeofday(&stK, NULL);
   	for (int it = 0; it < nt; it++){
	 	d_swap  = d_pp; 
	 	d_pp = d_p; 
	 	d_p = d_swap; 

	 	kernel_tapper<<<dimGridTaper, dimBlock>>>(nx,nz,nxbin,nzbin,d_p,d_pp,d_taperx,d_taperz);
	 	kernel_lap<<<dimGrid, dimBlock>>>(order,nx,nz,d_p,d_laplace,d_coefs_x,d_coefs_z);
	 	kernel_time<<<dimGrid, dimBlock>>>(nx,nz,d_p,d_pp,d_v2,d_laplace,dt2);
	 	kernel_src<<<dimGridSingle, dimBlock>>>(nz,d_pp,sx[is],sz,srce[it]); 
	 	kernel_upb<<<dimGridUpb, dimBlock>>>(order,nx,nz,nzbin,nt,d_pp,d_upb,it,0);
		cudaCheck();

     	if((it+1)%100 == 0){fprintf(stdout,"\r* it = %d / %d (%d%)",it+1,nt,(100*(it+1)/nt));fflush(stdout);}
     	// op_L_counter++;
 	} 
 	// Calc avg kernel time  
	gettimeofday(&etK, NULL);
	elapsed = ((etK.tv_sec - stK.tv_sec) * 1000000) + (etK.tv_usec - stK.tv_usec);
	kernelAVGTime += (elapsed*1.0);

	// start read time 
	gettimeofday(&stCR, NULL);
	if(propag == 5){
		float input[mtxBufferLength], output[mtxBufferLength];
		FILE *finput;
		finput = fopen("./input.bin", "wb");
		hipMemcpy(input, d_p, mtxBufferLength, hipMemcpyDeviceToHost);
		hipMemcpy(output, d_laplace, mtxBufferLength, hipMemcpyDeviceToHost);
		fwrite(input,sizeof(input),1,finput);
		printf("\n=== input: ===\n");
		for(int i = 1321; i < 1341; i++){
				printf("%.15f\n", input[i]);
		}
		printf("\n=== output: ===\n");
		for(int i = 1321; i < 1341; i++){
				printf("%.15f\n", output[i]);
		}
		fclose(finput);
	}
 	hipMemcpy(p[0], d_p, mtxBufferLength, hipMemcpyDeviceToHost);
 	hipMemcpy(pp[0], d_pp, mtxBufferLength, hipMemcpyDeviceToHost);
 	hipMemcpy(upb[0][0], d_upb, upbBufferLength, hipMemcpyDeviceToHost);
	// Calc avg read time  
	gettimeofday(&etCR, NULL);
	elapsed = ((etCR.tv_sec - stCR.tv_sec) * 1000000) + (etCR.tv_usec - stCR.tv_usec);
	rdAVGTime += (elapsed*1.0);
    rdTransferCnt++;	
 	gettimeofday(&et, NULL);
	elapsed = ((et.tv_sec - st.tv_sec) * 1000000) + (et.tv_usec - st.tv_usec);
	fwAVGTime += (elapsed*1.0);    
}

void fd_back(int order, float **p, float **pp, float **pr, float **ppr, float **v2, float ***upb,
   int nz, int nx, int nt, int is, int sz, int gz, float ***snaps, float **imloc, float **d_obs){
	int ix, iz, it, elapsed;
   	struct timeval st, et, stCR, etCR, stCW, etCW, stK, etK;
	//Start total time 
	gettimeofday(&st, NULL);
    
	dim3 dimGrid(gridx, gridz);	
  	dim3 dimGridTaper(gridx, gridBorder_z); 	
  	dim3 dimGridUpb(gridx,1); 
  	
  	dim3 dimBlock(sizeblock, sizeblock);
  	// Start write time	
	gettimeofday(&stCW, NULL);
	write_buffers(p,pp,v2,upb,taper_x, taper_z,d_obs,imloc,is,0);
	write_buffers(pr,ppr,v2,upb,taper_x,taper_z,d_obs,imloc,is,1);
	// Calc avg write time  
	gettimeofday(&etCW, NULL);
   	elapsed = ((etCW.tv_sec - stCW.tv_sec) * 1000000) + (etCW.tv_usec - stCW.tv_usec);
  	wrAVGTime += (elapsed*1.0);
	wrTransferCnt++; 
    // start Kernel time  
	gettimeofday(&stK, NULL);
   	for(it=0; it<nt; it++){
		gettimeofday(&etCW, NULL);
    	if(it==0 || it==1){
         for(ix=0; ix<nx; ix++){
            for(iz=0; iz<nz; iz++){
               pp[ix][iz] = snaps[1-it][ix][iz];                       
            }
         }
			hipMemcpy(d_pp, pp[0], mtxBufferLength, hipMemcpyHostToDevice);
      	}else{ 
		 	kernel_lap<<<dimGrid, dimBlock>>>(order,nx,nz,d_p,d_laplace,d_coefs_x,d_coefs_z);
		 	kernel_time<<<dimGrid, dimBlock>>>(nx,nz,d_p,d_pp,d_v2,d_laplace,dt2);
		 	kernel_upb<<<dimGridUpb, dimBlock>>>(order,nx,nz,nzbin,nt,d_pp,d_upb,it,1);
		}
			
		d_swap = d_pp; 
		d_pp = d_p; 
		d_p = d_swap;
		
		kernel_tapper<<<dimGridTaper, dimBlock>>>(nx,nz,nxbin,nzbin,d_pr,d_ppr,d_taperx,d_taperz);
		kernel_lap<<<dimGrid, dimBlock>>>(order,nx,nz,d_pr,d_laplace, d_coefs_x, d_coefs_z);
		kernel_time<<<dimGrid, dimBlock>>>(nx,nz,d_pr,d_ppr,d_v2,d_laplace,dt2);
		kernel_sism<<<dimGridUpb, dimBlock>>>(nx,nz,nxbin,nt,is,it,gz,d_sis,d_ppr);
		kernel_img<<<dimGrid, dimBlock>>>(nx,nz,nxbin,nzbin,d_img,d_p,d_ppr);
		
		d_swap = d_ppr; 
		d_ppr = d_pr; 
		d_pr = d_swap; 

		if((it+1)%100 == 0){fprintf(stdout,"\r* it = %d / %d (%d%)",it+1,nt,(100*(it+1)/nt));fflush(stdout);}
	}
	gettimeofday(&etK, NULL);
	elapsed = ((etK.tv_sec - stK.tv_sec) * 1000000) + (etK.tv_usec - stK.tv_usec);
	kernelAVGTime += (elapsed*1.0);
 	
 	gettimeofday(&stCR, NULL);
 	// Calc avg read time  
 	hipMemcpy(imloc[0], d_img, imgBufferLength, hipMemcpyDeviceToHost);
	gettimeofday(&etCR, NULL);
	elapsed = ((etCR.tv_sec - stCR.tv_sec) * 1000000) + (etCR.tv_usec - stCR.tv_usec);
	rdAVGTime += (elapsed*1.0);
	rdTransferCnt++;
	//bw avg time
	gettimeofday(&et, NULL);
	elapsed = ((et.tv_sec - st.tv_sec) * 1000000) + (et.tv_usec - st.tv_usec);
	bwAVGTime += (elapsed*1.0);
 	deviceAVGTime = fwAVGTime + bwAVGTime; 

}

