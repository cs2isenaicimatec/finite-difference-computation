#include "hip/hip_runtime.h"
/* Acoustic wavefield modeling using finite-difference method
Leonardo Gómez Bernal, Salvador BA, Brazil
August, 2016 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "fd.h"
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
extern "C" {
	#include "cwp.h"
	#include "su.h"
	#include "ptsrc.h"
	#include "taper.h"
}

char *sdoc[] = {	/* self documentation */
	" Seismic modeling using acoustic wave equation ",
	"				               ",
	NULL};
/* global variables */

/* file names */
char *tmpdir = NULL, *vpfile = NULL, *datfile = NULL, *vel_ext_file = NULL, file[100];

/* size */
int nz, nx, nt;
float dz, dx, dt;

/* adquisition geometry */
int ns, sz, fsx, ds, gz;

/* boundary */
int nxb, nzb, nxe, nze;
float fac;

/* propagation */
int order; 
float fpeak;

/* arrays */
int *sx;

/* prototypes */

int main (int argc, char **argv){
  hipProfilerStart();
		struct timeval st, et;
    int elapsed;
	float execTime;
	clock_t begin, end;
	long int time_spent;
 	begin = clock();
	gettimeofday(&st, NULL);
	
	/* model file and data pointers */
	FILE *fsource = NULL, *fvel_ext = NULL, *fd_obs = NULL, *fvp = NULL, *fsns = NULL,*fsns2 = NULL, *fsnr = NULL, *fimg = NULL, *flim = NULL, *fimg_lap = NULL;

	/* iteration variables */
	int iz, ix, it, is;

	/* auxiliar */
	int iss, rnd, vel_ext_flag=0;

	/* arrays */
	float *srce;
	float **vp = NULL, **vpe = NULL, **vpex = NULL;

	/* propagation variables */
	float **PP,**P,**PPR,**PR,**tmp;
	float ***swf, ***upb, ***snaps, **vel2, ***d_obs, ***vel_ext_rnd;
	float **imloc, **img, **img_lap;

	/* initialization admiting self documentation */
	initargs(argc, argv);
	requestdoc(1);

	/* read parameters */
	MUSTGETPARSTRING("tmpdir",&tmpdir);		// directory for data
	MUSTGETPARSTRING("vpfile",&vpfile);		// vp model
	MUSTGETPARSTRING("datfile",&datfile);	// observed data (seismogram)
	MUSTGETPARINT("nz",&nz); 				// number of samples in z
	MUSTGETPARINT("nx",&nx); 				// number of samples in x
	MUSTGETPARINT("nt",&nt); 				// number of time steps
	MUSTGETPARFLOAT("dz",&dz); 				// sampling interval in z
	MUSTGETPARFLOAT("dx",&dx); 				// sampling interval in x
	MUSTGETPARFLOAT("dt",&dt); 				// sampling interval in t
	MUSTGETPARFLOAT("fpeak",&fpeak); 		// souce peak frequency

	if(getparstring("vel_ext_file",&vel_ext_file)) vel_ext_flag = 1;
	if(!getparint("iss",&iss)) iss = 0;	 	// save snaps of this source
	if(!getparint("ns",&ns)) ns = 1;	 	// number of sources
	if(!getparint("sz",&sz)) sz = 0; 		// source depth
	if(!getparint("fsx",&fsx)) fsx = 0; 	// first source position
	if(!getparint("ds",&ds)) ds = 1; 		// source interval
	if(!getparint("gz",&gz)) gz = 0; 		// receivor depth

	if(!getparint("order",&order)) order = 8;	// FD order
	if(!getparint("nzb",&nzb)) nzb = 40;		// z border size
	if(!getparint("nxb",&nxb)) nxb = 40;		// x border size
	if(!getparfloat("fac",&fac)) fac = 0.7;		// damping factor
	// if(!getparint("rnd",&rnd)) rnd = 1;		    // random vel. border

	fprintf(stdout,"## vp = %s, d_obs = %s, vel_ext_file = %s, vel_ext_flag = %d \n",vpfile,datfile,vel_ext_file,vel_ext_flag);
	fprintf(stdout,"## nz = %d, nx = %d, nt = %d \n",nz,nx,nt);
	fprintf(stdout,"## dz = %f, dx = %f, dt = %f \n",dz,dx,dt);
	fprintf(stdout,"## ns = %d, sz = %d, fsx = %d, ds = %d, gz = %d \n",ns,sz,fsx,ds,gz);
	fprintf(stdout,"## order = %d, nzb = %d, nxb = %d, F = %f, rnd = %d \n",order,nzb,nxb,fac,rnd);
	/* create source vector  */
	srce = alloc1float(nt);
	ricker_wavelet(nt, dt, fpeak, srce);
	sx = alloc1int(ns);
	for(is=0; is<ns; is++){
		sx[is] = fsx + is*ds + nxb;
	}
	sz += nzb;
	gz += nzb;
	/* add boundary to models */
	nze = nz + 2 * nzb;
	nxe = nx + 2 * nxb;
	/*read randomic vel. models (per source) */
	if(vel_ext_flag){
		vel_ext_rnd = alloc3float(nze,nxe,ns);
		memset(**vel_ext_rnd,0,nze*nxe*ns*sizeof(float));
		fvel_ext = fopen(vel_ext_file,"r");
		fread(**vel_ext_rnd,sizeof(float),nze*nxe*ns,fvel_ext);
		fclose(fvel_ext);
	}

	/*read observed data (seism.) */
	d_obs = alloc3float(nt,nx,ns);
	memset(**d_obs,0,nt*nx*ns*sizeof(float));
	fd_obs = fopen(datfile,"r");
	fread(**d_obs,sizeof(float),nt*nx*ns,fd_obs);
	fclose(fd_obs);

	float **d_obs_aux=(float**)malloc(ns*sizeof(float*));
	for(int i=0; i<ns; i++) 
		d_obs_aux[i] = (float*)malloc((nt*nx)*sizeof(float)); 
	
	for(int i=0; i<ns; i++){
		for(int j=0; j<nx; j++){
			for(int k=0; k<nt; k++)
				d_obs_aux[i][j*nt+k] = d_obs[i][j][k]; 
		}
	}

	/* read parameter models */
	vp = alloc2float(nz,nx);
	memset(*vp,0,nz*nx*sizeof(float));
	fvp = fopen(vpfile,"r");
	fread(vp[0],sizeof(float),nz*nx,fvp);
	fclose(fvp);

	/* vp size estended to vpe */
	vpe = alloc2float(nze,nxe);
	vpex = vpe;

	for(ix=0; ix<nx; ix++){
		for(iz=0; iz<nz; iz++){
			vpe[ix+nxb][iz+nzb] = vp[ix][iz]; 
		}
	}

	/* allocate vel2 for vpe^2 */
	vel2 = alloc2float(nze,nxe);

	/* initialize wave propagation */
	fd_init(order,nxe,nze,nxb,nzb,nt,ns,fac,dx,dz,dt);
	taper_init(nxb,nzb,fac);

	PP = alloc2float(nze,nxe);
	P = alloc2float(nze,nxe);
	PPR = alloc2float(nze,nxe);
	PR = alloc2float(nze,nxe);
	upb = alloc3float(order/2,nxe,nt);
	// swf = alloc3float(nz,nx,nt);
	snaps = alloc3float(nze,nxe,2);
	imloc = alloc2float(nz,nx);
	img = alloc2float(nz,nx);
	img_lap = alloc2float(nz,nx);

	// fsns = fopen("output/dir.snaps","w");
	// fsns2 = fopen("output/dir.snaps_rec","w");
	// fsnr = fopen("output/dir.snapr","w");
	// fimg = fopen("output/dir.image","w");	
	// fimg_lap = fopen("output/dir.image_lap","w");

	char filepath [100];
	sprintf(filepath, "%s/dir.snaps", tmpdir);
	fsns = fopen(filepath,"w");
	sprintf(filepath, "%s/dir.snaps_rec", tmpdir);
	fsns2 = fopen(filepath,"w");
	sprintf(filepath, "%s/dir.snapr", tmpdir);
	fsnr = fopen(filepath,"w");
	sprintf(filepath, "%s/dir.image", tmpdir);
	fimg = fopen(filepath,"w");
	sprintf(filepath, "%s/dir.image_lap", tmpdir);	
	fimg_lap = fopen(filepath,"w");
	
	memset(*img,0,nz*nx*sizeof(float));
	memset(*img_lap,0,nz*nx*sizeof(float));

	for(is=0; is<ns; is++){
		fprintf(stdout,"** source %d, at (%d,%d) \n",is+1,sx[is]-nxb,sz-nzb);
		/* Calc (or load) velocity model border */
		if (vel_ext_flag){
			vpe = vel_ext_rnd[is];					// load hybrid border vpe from file
		}else{
			extendvel_linear(nx,nz,nxb,nzb,vpe); 	// hybrid border (linear randomic)
		}

		/* vel2 = vpe^2 */
		for(ix=0; ix<nx+2*nxb; ix++){
			for(iz=0; iz<nz+2*nzb; iz++){
				vel2[ix][iz] = vpe[ix][iz]*vpe[ix][iz];
			}
		}

		memset(*PP,0,nze*nxe*sizeof(float));
		memset(*P,0,nze*nxe*sizeof(float));
    
		hipProfilerStart();
		fd_forward(order,P,PP,vel2,upb,nze,nxe,nt,is,sz,sx,srce,is);
		fprintf(stdout,"\n");

		for(iz=0; iz<nze; iz++){
			for(ix=0; ix<nxe; ix++){
				snaps[0][ix][iz] = P[ix][iz];
				snaps[1][ix][iz] = PP[ix][iz];
			}
		}

		fprintf(stdout,"** backward propagation %d, at (%d,%d) \n",is+1,sx[is]-nxb,sz-nzb);

		memset(*PP,0,nze*nxe*sizeof(float));
		memset(*P,0,nze*nxe*sizeof(float));
		memset(*PPR,0,nze*nxe*sizeof(float));
		memset(*PR,0,nze*nxe*sizeof(float));
		memset(*imloc,0,nz*nx*sizeof(float));

		/* Reverse propagation */
		fd_back(order,P,PP,PR,PPR,vel2,upb,nze,nxe,nt,is,sz,gz,snaps,imloc,d_obs_aux);
		fprintf(stdout,"\n");
    hipProfilerStop();
		
		/* stack migrated images */
		for(iz=0; iz<nz; iz++){
			for(ix=0; ix<nx; ix++){
				img[ix][iz] += imloc[ix][iz];
			}
		}
	}
	
	hipProfilerStop();
	// hipDeviceReset();
#ifdef  PERF_COUNTERS
	fd_print_report(nxe, nze);
	gettimeofday(&et, NULL);
   	elapsed = ((et.tv_sec - st.tv_sec) * 1000000) + (et.tv_usec - st.tv_usec);
   	execTime += (elapsed*1.0);
   	printf("> Exec Time    = %.1f (s)\n",execTime/1000000.0);
	printf("> ================================================ \n\n");
#endif
	fwrite(*img,sizeof(float),nz*nx,fimg);

	fwrite(*img_lap,sizeof(float),nz*nx,fimg_lap);

	fclose(fsns);
	fclose(fsns2);
	fclose(fsnr);
	fclose(fimg);
	fclose(fimg_lap);

    /* release memory */
  fd_destroy();
	taper_destroy();
	free1int(sx);
	free1float(srce);
	free2float(vp);
	free2float(P);
	free2float(PP);
	free2float(PR);
	free2float(PPR);
	// free3float(swf);
	free3float(snaps);
	free2float(imloc);
	free2float(img);
	free2float(img_lap);
	free2float(vpex);
	free2float(vel2);
	free3float(upb);
	free3float(d_obs);
	if(vel_ext_flag) free3float(vel_ext_rnd);
	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	return(CWP_Exit());
}
