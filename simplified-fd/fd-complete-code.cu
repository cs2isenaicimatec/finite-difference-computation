#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>

void fd_init(int order, int nx, int nz, float dx, float dz);
void fd_init_cuda(int order, int nxe, int nze);
float *calc_coefs(int order);
static void makeo2 (float *coef,int order);
void read_input(char *file);
void *alloc1 (size_t n1, size_t size);
void **alloc2 (size_t n1, size_t n2, size_t size);
float *alloc1float(size_t n1);
float **alloc2float(size_t n1, size_t n2);
void free1 (void *p);
void free2 (void **p);
void free1float(float *p);
void free2float(float **p);

#define sizeblock 32
#define PI (3.141592653589793)

/* file names */
char *tmpdir = NULL, *vpfile = NULL, *datfile = NULL, *vel_ext_file = NULL, file[100];

float *d_p;
float *d_laplace, *d_coefs_x, *d_coefs_z;

size_t mtxBufferLength, coefsBufferLength;

int gridx, gridz;
/* size */
int nz, nx, nt;
float dz, dx, dt;

/* adquisition geometry */
int ns = -1, sz = -1, fsx = -1, ds = -1, gz = -1;

/* boundary */
int nxb = -1, nzb = -1, nxe, nze;
float fac = -1.0;

/* propagation */
int order = -1; 
float fpeak;

/* arrays */
int *sx;

/*aux*/
int iss = -1, rnd, vel_ext_flag=0;

static float dx2inv, dz2inv;
static float *coefs = NULL;
static float *coefs_z = NULL;
static float *coefs_x = NULL;

void read_input(char *file)
{
        FILE *fp;
        fp = fopen(file, "r");
        char *line = NULL;
        size_t len = 0;
        if (fp == NULL)
                exit(EXIT_FAILURE);
        while (getline(&line, &len, fp) != -1) {
                if(strstr(line,"tmpdir") != NULL)
                {
                        char *tok;
                        tok = strtok(line, "=");
                        tok = strtok(NULL,"=");
                        tok[strlen(tok) - 1] = '\0';
                        tmpdir = strdup(tok);
                }
                if(strstr(line,"datfile") != NULL)
                {
                        char *tok;
                        tok = strtok(line, "=");
                        tok = strtok(NULL,"=");
                        tok[strlen(tok) - 1] = '\0';
                        datfile = strdup(tok);
                }
                if(strstr(line,"vpfile") != NULL)
                {
                        char *tok;
                        tok = strtok(line, "=");
                        tok = strtok(NULL,"=");
                        tok[strlen(tok) - 1] = '\0';
                        vpfile = strdup(tok);
                }
                if(strstr(line,"vel_ext_file") != NULL)
                {
                        char *tok;
                        tok = strtok(line, "=");
                        tok = strtok(NULL,"=");
                        tok[strlen(tok) - 1] = '\0';
                        vel_ext_file = strdup(tok);
                        vel_ext_flag = 1;
                }
                if(strstr(line,"fpeak") != NULL)
                {
                        char *fpeak_char;
                        fpeak_char = strtok(line, "=");
                        fpeak_char = strtok(NULL,"=");
                        fpeak = atof(fpeak_char);
                }
                if(strstr(line,"nt") != NULL)
                {
                        char *nt_char;
                        nt_char = strtok(line, "=");
                        nt_char = strtok(NULL,"=");
                        nt = atoi(nt_char);
                }
                if(strstr(line,"dt") != NULL)
                {
                        char *dt_char;
                        dt_char = strtok(line, "=");
                        dt_char = strtok(NULL,"=");
                        dt = atof(dt_char);
                }
                if(strstr(line,"ns") != NULL)
                {
                        char *ns_char;
                        ns_char = strtok(line, "=");
                        ns_char = strtok(NULL,"=");
                        ns = atoi(ns_char);
                }
                if(strstr(line,"iss") != NULL)
                {
                        char *iss_char;
                        iss_char = strtok(line, "=");
                        iss_char = strtok(NULL,"=");
                        iss = atoi(iss_char);
                }
                if(strstr(line,"sz") != NULL)
                {
                        char *sz_char;
                        sz_char = strtok(line, "=");
                        sz_char = strtok(NULL,"=");
                        sz = atoi(sz_char);
                }
                if(strstr(line,"fsx") != NULL)
                {
                        char *fsx_char;
                        fsx_char = strtok(line, "=");
                        fsx_char = strtok(NULL,"=");
                        fsx = atoi(fsx_char);
                }
                if(strstr(line,"ds") != NULL)
                {
                        char *ds_char;
                        ds_char = strtok(line, "=");
                        ds_char = strtok(NULL,"=");
                        ds = atoi(ds_char);
                }
                if(strstr(line,"gz") != NULL)
                {
                        char *gz_char;
                        gz_char = strtok(line, "=");
                        gz_char = strtok(NULL,"=");
                        gz = atoi(gz_char);
                }
                if(strstr(line,"nzb") != NULL)
                {
                        char *nzb_char;
                        nzb_char = strtok(line, "=");
                        nzb_char = strtok(NULL,"=");
                        nzb = atoi(nzb_char);
                }
                if(strstr(line,"nxb") != NULL)
                {
                        char *nxb_char;
                        nxb_char = strtok(line, "=");
                        nxb_char = strtok(NULL,"=");
                        nxb = atoi(nxb_char);
                }
                if(strstr(line,"rnd") != NULL)
                {
                        char *rnd_char;
                        rnd_char = strtok(line, "=");
                        rnd_char = strtok(NULL,"=");
                        rnd = atoi(rnd_char);
                }
                if(strstr(line,"nz") != NULL)
                {
                        char *nz_char;
                        nz_char = strtok(line, "=");
                        if (strlen(nz_char) <= 2)
                        {
                                nz_char = strtok(NULL,"=");
                                nz = atoi(nz_char);
                        }
                }
                if(strstr(line,"nx") != NULL)
                {
                        char *nx_char;
                        nx_char = strtok(line, "=");
                        if (strlen(nx_char) <= 2)
                        {
                                nx_char = strtok(NULL,"=");
                                nx = atoi(nx_char);
                        }
                }
                if(strstr(line,"dz") != NULL)
                {
                        char *dz_char;
                        dz_char = strtok(line, "=");
                        dz_char = strtok(NULL,"=");
                        dz = atof(dz_char);
                }
                if(strstr(line,"dx") != NULL)
                {
                        char *dx_char;
                        dx_char = strtok(line, "=");
                        dx_char = strtok(NULL,"=");
                        dx = atof(dx_char);
                }
                if(strstr(line,"fac") != NULL)
                {
                        char *fac_char;
                        fac_char = strtok(line, "=");
                        fac_char = strtok(NULL,"=");
                        fac = atof(fac_char);
                }
                if(strstr(line,"order") != NULL)
                {
                        char *order_char;
                        order_char = strtok(line, "=");
                        order_char = strtok(NULL,"=");
                        order = atoi(order_char);
                }
        }
	if(iss == -1 ) iss = 0;	 	// save snaps of this source
	if(ns == -1) ns = 1;	 	// number of sources
	if(sz == -1) sz = 0; 		// source depth
	if(fsx == -1) fsx = 0; 	// first source position
	if(ds == -1) ds = 1; 		// source interval
	if(gz == -1) gz = 0; 		// receivor depth
	if(order == -1) order = 8;	// FD order
	if(nzb == -1) nzb = 40;		// z border size
	if(nxb == -1) nxb = 40;		// x border size
	if(fac == -1.0) fac = 0.7;	
}

__global__ void kernel_lap(int order, int nx, int nz, float * __restrict__ p, float * __restrict__ lap, float * __restrict__ coefsx, float * __restrict__ coefsz)
{
        int half_order=order/2;
        int i =  half_order + blockIdx.x * blockDim.x + threadIdx.x; // Global row index
        int j =  half_order + blockIdx.y * blockDim.y + threadIdx.y; // Global column index
        int mult = i*nz;
        int aux;
        float acmx = 0, acmz = 0;

        if(i<nx - half_order)
        {
                if(j<nz - half_order)
                {
                        for(int io=0;io<=order;io++)
                        {
                                aux = io-half_order;
                                acmz += p[mult + j+aux]*coefsz[io];
                                acmx += p[(i+aux)*nz + j]*coefsx[io];
                        }
                        lap[mult +j] = acmz + acmx;
                        acmx = 0.0;
                        acmz = 0.0;
                }
        }

}

float *calc_coefs(int order)
{
        float *coef;

        coef = (float *)calloc(order+1,sizeof(float));
        switch(order)
        {
                case 2:
                        coef[0] = 1.;
                        coef[1] = -2.;
                        coef[2] = 1.;
                        break;
                case 4:
                        coef[0] = -1./12.;
                        coef[1] = 4./3.;
                        coef[2] = -5./2.;
                        coef[3] = 4./3.;
                        coef[4] = -1./12.;
                        break;
                case 6:
                        coef[0] = 1./90.;
                        coef[1] = -3./20.;
                        coef[2] = 3./2.;
                        coef[3] = -49./18.;
                        coef[4] = 3./2.;
                        coef[5] = -3./20.;
                        coef[6] = 1./90.;
                        break;
                case 8:

                        coef[0] = -1./560.;
                        coef[1] = 8./315.;
                        coef[2] = -1./5.;
                        coef[3] = 8./5.;
                        coef[4] = -205./72.;
                        coef[5] = 8./5.;
                        coef[6] = -1./5.;
                        coef[7] = 8./315.;
                        coef[8] = -1./560.;
                        break;
                default:
                        makeo2(coef,order);
        }

        return coef;
}

static void makeo2 (float *coef,int order)
{
        float h_beta, alpha1=0.0;
        float alpha2=0.0;
        float  central_term=0.0;
        float coef_filt=0;
        float arg=0.0;
        float  coef_wind=0.0;
        int msign,ix;

        float alpha = .54;
        float beta = 6.;
        h_beta = 0.5*beta;
        alpha1=2.*alpha-1.0;
        alpha2=2.*(1.0-alpha);
        central_term=0.0;

        msign=-1;

        for (ix=1; ix <= order/2; ix++){
                msign=-msign ;
                coef_filt = (2.*msign)/(ix*ix);
                arg = PI*ix/(2.*(order/2+2));
                coef_wind=pow((alpha1+alpha2*cos(arg)*cos(arg)),h_beta);
                coef[order/2+ix] = coef_filt*coef_wind;
                central_term = central_term + coef[order/2+ix];
                coef[order/2-ix] = coef[order/2+ix];
        }

        coef[order/2]  = -2.*central_term;

        return;
}

void *alloc1 (size_t n1, size_t size)
{
	void *p;

	if ((p=malloc(n1*size))==NULL)
		return NULL;
	return p;
}

void **alloc2 (size_t n1, size_t n2, size_t size)
{
	size_t i2;
	void **p;

	if ((p=(void**)malloc(n2*sizeof(void*)))==NULL) 
		return NULL;
	if ((p[0]=(void*)malloc(n2*n1*size))==NULL) {
		free(p);
		return NULL;
	}
	for (i2=0; i2<n2; i2++)
		p[i2] = (char*)p[0]+size*n1*i2;
	return p;
}

float *alloc1float(size_t n1)
{
	return (float*)alloc1(n1,sizeof(float));
}

float **alloc2float(size_t n1, size_t n2)
{
	return (float**)alloc2(n1,n2,sizeof(float));
}

void free1 (void *p)
{
	free(p);
}

void free2 (void **p)
{
	free(p[0]);
	free(p);
}

void free1float(float *p)
{
	free1(p);
}

void free2float(float **p)
{
	free2((void**)p);
}

void fd_init_cuda(int order, int nxe, int nze)
{
        mtxBufferLength = (nxe*nze)*sizeof(float);
        coefsBufferLength = (order+1)*sizeof(float);

        // Create a Device pointers
        hipMalloc(&d_p, mtxBufferLength);
        hipMalloc(&d_laplace, mtxBufferLength);
        hipMalloc(&d_coefs_x, coefsBufferLength);
        hipMalloc(&d_coefs_z, coefsBufferLength);

        int div_x, div_z;
        // Set a Grid for the execution on the device
        int tx = ((nxe - 1) / 32 + 1) * 32;
        int tz = ((nze - 1) / 32 + 1) * 32;

        div_x = (float) tx/(float) sizeblock;
        div_z = (float) tz/(float) sizeblock;

        gridx = (int) ceil(div_x);
        gridz = (int) ceil(div_z);
}

void fd_init(int order, int nx, int nz, float dx, float dz)
{
        int io;
        dx2inv = (1./dx)*(1./dx);
        dz2inv = (1./dz)*(1./dz);

        coefs = calc_coefs(order);

        coefs_z = calc_coefs(order);
        coefs_x = calc_coefs(order);

        // pre calc coefs 8 d2 inv
        for (io = 0; io <= order; io++)
        {
                coefs_z[io] = dz2inv * coefs[io];
                coefs_x[io] = dx2inv * coefs[io];
        }

        fd_init_cuda(order,nx,nz);

        return;
}

int main (int argc, char **argv)
{
        /* model file and data pointers */
	FILE *fsource = NULL, *fvel_ext = NULL, *fd_obs = NULL, *fvp = NULL, *fsns = NULL,*fsns2 = NULL, *fsnr = NULL, *fimg = NULL, *flim = NULL, *fimg_lap = NULL;

	/* iteration variables */
	int iz, ix, it, is;

	/* arrays */
	float *srce;
	float **vp = NULL, **vpe = NULL, **vpex = NULL;

	/* propagation variables */
	float **PP,**P,**PPR,**PR,**tmp;
	float ***swf, ***upb, ***snaps, **vel2, ***d_obs, ***vel_ext_rnd;
	float **imloc, **img, **img_lap;
        read_input(argv[1]);

        printf("## vp = %s, d_obs = %s, vel_ext_file = %s, vel_ext_flag = %d \n",vpfile,datfile,vel_ext_file,vel_ext_flag);
	printf("## nz = %d, nx = %d, nt = %d \n",nz,nx,nt);
	printf("## dz = %f, dx = %f, dt = %f \n",dz,dx,dt);
	printf("## ns = %d, sz = %d, fsx = %d, ds = %d, gz = %d \n",ns,sz,fsx,ds,gz);
	printf("## order = %d, nzb = %d, nxb = %d, F = %f, rnd = %d \n",order,nzb,nxb,fac,rnd);

        // nxe = nx + 2 * nxb;
        // nze = nz + 2 * nzb;
        // // inicialização
        // fd_init(order,nxe,nze,dx,dz);

        // dim3 dimGrid(gridx, gridz);
        // dim3 dimBlock(sizeblock, sizeblock);
        // FILE *finput;
        // float *input_data;

        // if((finput = fopen(path_file, "rb")) == NULL)
        //         printf("Unable to open file!\n");
        // else
        //         printf("Opened input successfully for read.\n");
        
        // input_data = (float*)malloc(mtxBufferLength);
        // if(!input_data)
        //         printf("input Memory allocation error!\n");
        // else 
        //         printf("input Memory allocation successful.\n");

        // memset(input_data, 0, mtxBufferLength);
        
        // if( fread(input_data, sizeof(float), nze*nxe, finput) != nze*nxe)
        //         printf("input Read error!\n");
        
        // else 
        //         printf("input Read was successful.\n");
        // fclose(finput);

        // // utilização do kernel
        // cudaMemcpy(d_p, input_data, mtxBufferLength, cudaMemcpyHostToDevice);
        // cudaMemcpy(d_coefs_x, coefs_x, coefsBufferLength, cudaMemcpyHostToDevice);
        // cudaMemcpy(d_coefs_z, coefs_z, coefsBufferLength, cudaMemcpyHostToDevice);

        // kernel_lap<<<dimGrid, dimBlock>>>(order,nxe,nze,d_p,d_laplace,d_coefs_x,d_coefs_z);

        // float *output_data;
        // output_data = (float*)malloc(mtxBufferLength);
        // if(!output_data)
        //         printf("output Memory allocation error!\n");
        // else 
        //         printf("output Memory allocation successful.\n");
        // memset(output_data, 0, mtxBufferLength);
        // cudaMemcpy(output_data, d_laplace, mtxBufferLength, cudaMemcpyDeviceToHost);

        // // salvando a saída
        // FILE *foutput;
        // if((foutput = fopen("output_cuda.bin", "wb")) == NULL)
        //         printf("Unable to open file!\n");
        // else
        //         printf("Opened output successfully for write.\n");
        
        // if( fwrite(output_data, sizeof(float), nze*nxe, foutput) != nze*nxe)
        //         printf("output Write error!\n");
        
        // else 
        //         printf("output Write was successful.\n");
        // fclose(foutput);

        // // free memory device
        // free(input_data);
        // free(output_data);
        // cudaFree(d_p);
        // cudaFree(d_laplace);
        // cudaFree(d_coefs_x);
        // cudaFree(d_coefs_z);
        return 0;
}

//==============================================================================================================================================
/* Acoustic wavefield modeling using finite-difference method
Leonardo Gómez Bernal, Salvador BA, Brazil
August, 2016 */

// #include <stdio.h>
// #include <cuda.h>
// #include <time.h>
// #include "fd.h"
// #include <sys/time.h>
// #include <cuda_runtime.h>
// #include <cuda_profiler_api.h>
// extern "C" {
// 	#include "cwp.h"
// 	#include "su.h"
// 	#include "ptsrc.h"
// 	#include "taper.h"
// }

// char *sdoc[] = {	/* self documentation */
// 	" Seismic modeling using acoustic wave equation ",
// 	"				               ",
// 	NULL};
// /* global variables */


// /* prototypes */

// int main (int argc, char **argv){
//   cudaProfilerStart();
// 		struct timeval st, et;
//     int elapsed;
// 	float execTime;
// 	clock_t begin, end;
// 	long int time_spent;
//  	begin = clock();
// 	gettimeofday(&st, NULL);
	
	

// 	/* initialization admiting self documentation */
// 	initargs(argc, argv);
// 	requestdoc(1);

// 	/* read parameters */
// 	MUSTGETPARSTRING("tmpdir",&tmpdir);		// directory for data
// 	MUSTGETPARSTRING("vpfile",&vpfile);		// vp model
// 	MUSTGETPARSTRING("datfile",&datfile);	// observed data (seismogram)
// 	MUSTGETPARINT("nz",&nz); 				// number of samples in z
// 	MUSTGETPARINT("nx",&nx); 				// number of samples in x
// 	MUSTGETPARINT("nt",&nt); 				// number of time steps
// 	MUSTGETPARFLOAT("dz",&dz); 				// sampling interval in z
// 	MUSTGETPARFLOAT("dx",&dx); 				// sampling interval in x
// 	MUSTGETPARFLOAT("dt",&dt); 				// sampling interval in t
// 	MUSTGETPARFLOAT("fpeak",&fpeak); 		// souce peak frequency

// 	if(getparstring("vel_ext_file",&vel_ext_file)) vel_ext_flag = 1;
// 	if(!getparint("iss",&iss)) iss = 0;	 	// save snaps of this source
// 	if(!getparint("ns",&ns)) ns = 1;	 	// number of sources
// 	if(!getparint("sz",&sz)) sz = 0; 		// source depth
// 	if(!getparint("fsx",&fsx)) fsx = 0; 	// first source position
// 	if(!getparint("ds",&ds)) ds = 1; 		// source interval
// 	if(!getparint("gz",&gz)) gz = 0; 		// receivor depth

// 	if(!getparint("order",&order)) order = 8;	// FD order
// 	if(!getparint("nzb",&nzb)) nzb = 40;		// z border size
// 	if(!getparint("nxb",&nxb)) nxb = 40;		// x border size
// 	if(!getparfloat("fac",&fac)) fac = 0.7;		// damping factor
// 	// if(!getparint("rnd",&rnd)) rnd = 1;		    // random vel. border

// 	fprintf(stdout,"## vp = %s, d_obs = %s, vel_ext_file = %s, vel_ext_flag = %d \n",vpfile,datfile,vel_ext_file,vel_ext_flag);
// 	fprintf(stdout,"## nz = %d, nx = %d, nt = %d \n",nz,nx,nt);
// 	fprintf(stdout,"## dz = %f, dx = %f, dt = %f \n",dz,dx,dt);
// 	fprintf(stdout,"## ns = %d, sz = %d, fsx = %d, ds = %d, gz = %d \n",ns,sz,fsx,ds,gz);
// 	fprintf(stdout,"## order = %d, nzb = %d, nxb = %d, F = %f, rnd = %d \n",order,nzb,nxb,fac,rnd);
// 	/* create source vector  */
// 	srce = alloc1float(nt);
// 	ricker_wavelet(nt, dt, fpeak, srce);
// 	sx = alloc1int(ns);
// 	for(is=0; is<ns; is++){
// 		sx[is] = fsx + is*ds + nxb;
// 	}
// 	sz += nzb;
// 	gz += nzb;
// 	/* add boundary to models */
// 	nze = nz + 2 * nzb;
// 	nxe = nx + 2 * nxb;
// 	/*read randomic vel. models (per source) */
// 	if(vel_ext_flag){
// 		vel_ext_rnd = alloc3float(nze,nxe,ns);
// 		memset(**vel_ext_rnd,0,nze*nxe*ns*sizeof(float));
// 		fvel_ext = fopen(vel_ext_file,"r");
// 		fread(**vel_ext_rnd,sizeof(float),nze*nxe*ns,fvel_ext);
// 		fclose(fvel_ext);
// 	}

// 	/*read observed data (seism.) */
// 	d_obs = alloc3float(nt,nx,ns);
// 	memset(**d_obs,0,nt*nx*ns*sizeof(float));
// 	fd_obs = fopen(datfile,"r");
// 	fread(**d_obs,sizeof(float),nt*nx*ns,fd_obs);
// 	fclose(fd_obs);

// 	float **d_obs_aux=(float**)malloc(ns*sizeof(float*));
// 	for(int i=0; i<ns; i++) 
// 		d_obs_aux[i] = (float*)malloc((nt*nx)*sizeof(float)); 
	
// 	for(int i=0; i<ns; i++){
// 		for(int j=0; j<nx; j++){
// 			for(int k=0; k<nt; k++)
// 				d_obs_aux[i][j*nt+k] = d_obs[i][j][k]; 
// 		}
// 	}

// 	/* read parameter models */
// 	vp = alloc2float(nz,nx);
// 	memset(*vp,0,nz*nx*sizeof(float));
// 	fvp = fopen(vpfile,"r");
// 	fread(vp[0],sizeof(float),nz*nx,fvp);
// 	fclose(fvp);

// 	/* vp size estended to vpe */
// 	vpe = alloc2float(nze,nxe);
// 	vpex = vpe;

// 	for(ix=0; ix<nx; ix++){
// 		for(iz=0; iz<nz; iz++){
// 			vpe[ix+nxb][iz+nzb] = vp[ix][iz]; 
// 		}
// 	}

// 	/* allocate vel2 for vpe^2 */
// 	vel2 = alloc2float(nze,nxe);

// 	/* initialize wave propagation */
// 	fd_init(order,nxe,nze,nxb,nzb,nt,ns,fac,dx,dz,dt);
// 	taper_init(nxb,nzb,fac);

// 	PP = alloc2float(nze,nxe);
// 	P = alloc2float(nze,nxe);
// 	PPR = alloc2float(nze,nxe);
// 	PR = alloc2float(nze,nxe);
// 	upb = alloc3float(order/2,nxe,nt);
// 	// swf = alloc3float(nz,nx,nt);
// 	snaps = alloc3float(nze,nxe,2);
// 	imloc = alloc2float(nz,nx);
// 	img = alloc2float(nz,nx);
// 	img_lap = alloc2float(nz,nx);

// 	// fsns = fopen("output/dir.snaps","w");
// 	// fsns2 = fopen("output/dir.snaps_rec","w");
// 	// fsnr = fopen("output/dir.snapr","w");
// 	// fimg = fopen("output/dir.image","w");	
// 	// fimg_lap = fopen("output/dir.image_lap","w");

// 	char filepath [100];
// 	sprintf(filepath, "%s/dir.snaps", tmpdir);
// 	fsns = fopen(filepath,"w");
// 	sprintf(filepath, "%s/dir.snaps_rec", tmpdir);
// 	fsns2 = fopen(filepath,"w");
// 	sprintf(filepath, "%s/dir.snapr", tmpdir);
// 	fsnr = fopen(filepath,"w");
// 	sprintf(filepath, "%s/dir.image", tmpdir);
// 	fimg = fopen(filepath,"w");
// 	sprintf(filepath, "%s/dir.image_lap", tmpdir);	
// 	fimg_lap = fopen(filepath,"w");
	
// 	memset(*img,0,nz*nx*sizeof(float));
// 	memset(*img_lap,0,nz*nx*sizeof(float));

// 	for(is=0; is<ns; is++){
// 		fprintf(stdout,"** source %d, at (%d,%d) \n",is+1,sx[is]-nxb,sz-nzb);
// 		/* Calc (or load) velocity model border */
// 		if (vel_ext_flag){
// 			vpe = vel_ext_rnd[is];					// load hybrid border vpe from file
// 		}else{
// 			extendvel_linear(nx,nz,nxb,nzb,vpe); 	// hybrid border (linear randomic)
// 		}

// 		/* vel2 = vpe^2 */
// 		for(ix=0; ix<nx+2*nxb; ix++){
// 			for(iz=0; iz<nz+2*nzb; iz++){
// 				vel2[ix][iz] = vpe[ix][iz]*vpe[ix][iz];
// 			}
// 		}

// 		memset(*PP,0,nze*nxe*sizeof(float));
// 		memset(*P,0,nze*nxe*sizeof(float));
    
// 		cudaProfilerStart();
// 		fd_forward(order,P,PP,vel2,upb,nze,nxe,nt,is,sz,sx,srce, is);
// 		fprintf(stdout,"\n");

// 		for(iz=0; iz<nze; iz++){
// 			for(ix=0; ix<nxe; ix++){
// 				snaps[0][ix][iz] = P[ix][iz];
// 				snaps[1][ix][iz] = PP[ix][iz];
// 			}
// 		}

// 		fprintf(stdout,"** backward propagation %d, at (%d,%d) \n",is+1,sx[is]-nxb,sz-nzb);

// 		memset(*PP,0,nze*nxe*sizeof(float));
// 		memset(*P,0,nze*nxe*sizeof(float));
// 		memset(*PPR,0,nze*nxe*sizeof(float));
// 		memset(*PR,0,nze*nxe*sizeof(float));
// 		memset(*imloc,0,nz*nx*sizeof(float));

// 		/* Reverse propagation */
// 		fd_back(order,P,PP,PR,PPR,vel2,upb,nze,nxe,nt,is,sz,gz,snaps,imloc,d_obs_aux);
// 		fprintf(stdout,"\n");
//     cudaProfilerStop();
		
// 		/* stack migrated images */
// 		for(iz=0; iz<nz; iz++){
// 			for(ix=0; ix<nx; ix++){
// 				img[ix][iz] += imloc[ix][iz];
// 			}
// 		}
// 	}
	
// 	cudaProfilerStop();
// 	// cudaDeviceReset();
// #ifdef  PERF_COUNTERS
// 	fd_print_report(nxe, nze);
// 	gettimeofday(&et, NULL);
//    	elapsed = ((et.tv_sec - st.tv_sec) * 1000000) + (et.tv_usec - st.tv_usec);
//    	execTime += (elapsed*1.0);
//    	printf("> Exec Time    = %.1f (s)\n",execTime/1000000.0);
// 	printf("> ================================================ \n\n");
// #endif
// 	fwrite(*img,sizeof(float),nz*nx,fimg);

// 	fwrite(*img_lap,sizeof(float),nz*nx,fimg_lap);

// 	fclose(fsns);
// 	fclose(fsns2);
// 	fclose(fsnr);
// 	fclose(fimg);
// 	fclose(fimg_lap);

//     /* release memory */
//   fd_destroy();
// 	taper_destroy();
// 	free1int(sx);
// 	free1float(srce);
// 	free2float(vp);
// 	free2float(P);
// 	free2float(PP);
// 	free2float(PR);
// 	free2float(PPR);
// 	// free3float(swf);
// 	free3float(snaps);
// 	free2float(imloc);
// 	free2float(img);
// 	free2float(img_lap);
// 	free2float(vpex);
// 	free2float(vel2);
// 	free3float(upb);
// 	free3float(d_obs);
// 	if(vel_ext_flag) free3float(vel_ext_rnd);
// 	end = clock();
// 	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
// 	return(CWP_Exit());
// }
