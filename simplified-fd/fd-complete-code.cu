#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>

void fd_init(int order, int nx, int nz, int nxb, int nzb, int nt, int ns, float fac, float dx, float dz, float dt);
void fd_init_cuda(int order, int nxe, int nze, int nxb, int nzb, int nt, int ns, float fac);
float *calc_coefs(int order);
static void makeo2 (float *coef,int order);
void read_input(char *file);
void *alloc1 (size_t n1, size_t size);
void **alloc2 (size_t n1, size_t n2, size_t size);
float *alloc1float(size_t n1);
float **alloc2float(size_t n1, size_t n2);
void free1 (void *p);
void free2 (void **p);
void free1float(float *p);
void free2float(float **p);

#define sizeblock 32
#define PI (3.141592653589793)

/* file names */
char *tmpdir = NULL, *vpfile = NULL, *datfile = NULL, *vel_ext_file = NULL;
/* size */
int nz, nx, nt;
float dz, dx, dt;

/* adquisition geometry */
int ns = -1, sz = -1, fsx = -1, ds = -1, gz = -1;

/* boundary */
int nxb = -1, nzb = -1, nxe, nze;
float fac = -1.0;

/* propagation */
int order = -1; 
float fpeak;

/* arrays */
int *sx;

/*aux*/
int iss = -1, rnd, vel_ext_flag=0;

float *d_p, *d_pr, *d_pp, *d_ppr, *d_swap;
float *d_laplace, *d_v2, *d_coefs_x, *d_coefs_z;
float *d_taperx, *d_taperz, *d_upb, *d_sis, *d_img;

size_t mtxBufferLength, brdBufferLength;
size_t imgBufferLength, obsBufferLength;
size_t coefsBufferLength, upbBufferLength;

float *taper_x, *taper_z;
int nxbin, nzbin;

int gridx, gridz, gridupb;
int gridBorder_x, gridBorder_z;

static float dx2inv,dz2inv,dt2;
static float **laplace = NULL;
static float *coefs = NULL;
static float *coefs_z = NULL;
static float *coefs_x = NULL;

void read_input(char *file)
{
        FILE *fp;
        fp = fopen(file, "r");
        char *line = NULL;
        size_t len = 0;
        if (fp == NULL)
                exit(EXIT_FAILURE);
        while (getline(&line, &len, fp) != -1) {
                if(strstr(line,"tmpdir") != NULL)
                {
                        char *tok;
                        tok = strtok(line, "=");
                        tok = strtok(NULL,"=");
                        tok[strlen(tok) - 1] = '\0';
                        tmpdir = strdup(tok);
                }
                if(strstr(line,"datfile") != NULL)
                {
                        char *tok;
                        tok = strtok(line, "=");
                        tok = strtok(NULL,"=");
                        tok[strlen(tok) - 1] = '\0';
                        datfile = strdup(tok);
                }
                if(strstr(line,"vpfile") != NULL)
                {
                        char *tok;
                        tok = strtok(line, "=");
                        tok = strtok(NULL,"=");
                        tok[strlen(tok) - 1] = '\0';
                        vpfile = strdup(tok);
                }
                if(strstr(line,"vel_ext_file") != NULL)
                {
                        char *tok;
                        tok = strtok(line, "=");
                        tok = strtok(NULL,"=");
                        tok[strlen(tok) - 1] = '\0';
                        vel_ext_file = strdup(tok);
                        vel_ext_flag = 1;
                }
                if(strstr(line,"fpeak") != NULL)
                {
                        char *fpeak_char;
                        fpeak_char = strtok(line, "=");
                        fpeak_char = strtok(NULL,"=");
                        fpeak = atof(fpeak_char);
                }
                if(strstr(line,"nt") != NULL)
                {
                        char *nt_char;
                        nt_char = strtok(line, "=");
                        nt_char = strtok(NULL,"=");
                        nt = atoi(nt_char);
                }
                if(strstr(line,"dt") != NULL)
                {
                        char *dt_char;
                        dt_char = strtok(line, "=");
                        dt_char = strtok(NULL,"=");
                        dt = atof(dt_char);
                }
                if(strstr(line,"ns") != NULL)
                {
                        char *ns_char;
                        ns_char = strtok(line, "=");
                        ns_char = strtok(NULL,"=");
                        ns = atoi(ns_char);
                }
                if(strstr(line,"iss") != NULL)
                {
                        char *iss_char;
                        iss_char = strtok(line, "=");
                        iss_char = strtok(NULL,"=");
                        iss = atoi(iss_char);
                }
                if(strstr(line,"sz") != NULL)
                {
                        char *sz_char;
                        sz_char = strtok(line, "=");
                        sz_char = strtok(NULL,"=");
                        sz = atoi(sz_char);
                }
                if(strstr(line,"fsx") != NULL)
                {
                        char *fsx_char;
                        fsx_char = strtok(line, "=");
                        fsx_char = strtok(NULL,"=");
                        fsx = atoi(fsx_char);
                }
                if(strstr(line,"ds") != NULL)
                {
                        char *ds_char;
                        ds_char = strtok(line, "=");
                        ds_char = strtok(NULL,"=");
                        ds = atoi(ds_char);
                }
                if(strstr(line,"gz") != NULL)
                {
                        char *gz_char;
                        gz_char = strtok(line, "=");
                        gz_char = strtok(NULL,"=");
                        gz = atoi(gz_char);
                }
                if(strstr(line,"nzb") != NULL)
                {
                        char *nzb_char;
                        nzb_char = strtok(line, "=");
                        nzb_char = strtok(NULL,"=");
                        nzb = atoi(nzb_char);
                }
                if(strstr(line,"nxb") != NULL)
                {
                        char *nxb_char;
                        nxb_char = strtok(line, "=");
                        nxb_char = strtok(NULL,"=");
                        nxb = atoi(nxb_char);
                }
                if(strstr(line,"rnd") != NULL)
                {
                        char *rnd_char;
                        rnd_char = strtok(line, "=");
                        rnd_char = strtok(NULL,"=");
                        rnd = atoi(rnd_char);
                }
                if(strstr(line,"nz") != NULL)
                {
                        char *nz_char;
                        nz_char = strtok(line, "=");
                        if (strlen(nz_char) <= 2)
                        {
                                nz_char = strtok(NULL,"=");
                                nz = atoi(nz_char);
                        }
                }
                if(strstr(line,"nx") != NULL)
                {
                        char *nx_char;
                        nx_char = strtok(line, "=");
                        if (strlen(nx_char) <= 2)
                        {
                                nx_char = strtok(NULL,"=");
                                nx = atoi(nx_char);
                        }
                }
                if(strstr(line,"dz") != NULL)
                {
                        char *dz_char;
                        dz_char = strtok(line, "=");
                        dz_char = strtok(NULL,"=");
                        dz = atof(dz_char);
                }
                if(strstr(line,"dx") != NULL)
                {
                        char *dx_char;
                        dx_char = strtok(line, "=");
                        dx_char = strtok(NULL,"=");
                        dx = atof(dx_char);
                }
                if(strstr(line,"fac") != NULL)
                {
                        char *fac_char;
                        fac_char = strtok(line, "=");
                        fac_char = strtok(NULL,"=");
                        fac = atof(fac_char);
                }
                if(strstr(line,"order") != NULL)
                {
                        char *order_char;
                        order_char = strtok(line, "=");
                        order_char = strtok(NULL,"=");
                        order = atoi(order_char);
                }
        }
        free(line);
	if(iss == -1 ) iss = 0;	 	// save snaps of this source
	if(ns == -1) ns = 1;	 	// number of sources
	if(sz == -1) sz = 0; 		// source depth
	if(fsx == -1) fsx = 0; 	// first source position
	if(ds == -1) ds = 1; 		// source interval
	if(gz == -1) gz = 0; 		// receivor depth
	if(order == -1) order = 8;	// FD order
	if(nzb == -1) nzb = 40;		// z border size
	if(nxb == -1) nxb = 40;		// x border size
	if(fac == -1.0) fac = 0.7;	
}
// ============================ Kernels ============================
__global__ void kernel_lap(int order, int nx, int nz, float * __restrict__ p, float * __restrict__ lap, float * __restrict__ coefsx, float * __restrict__ coefsz)
{
        int half_order=order/2;
        int i =  half_order + blockIdx.x * blockDim.x + threadIdx.x; // Global row index
        int j =  half_order + blockIdx.y * blockDim.y + threadIdx.y; // Global column index
        int mult = i*nz;
        int aux;
        float acmx = 0, acmz = 0;

        if(i<nx - half_order)
        {
                if(j<nz - half_order)
                {
                        for(int io=0;io<=order;io++)
                        {
                                aux = io-half_order;
                                acmz += p[mult + j+aux]*coefsz[io];
                                acmx += p[(i+aux)*nz + j]*coefsx[io];
                        }
                        lap[mult +j] = acmz + acmx;
                        acmx = 0.0;
                        acmz = 0.0;
                }
        }

}

__global__ void kernel_lap(int order, int nx, int nz, float * __restrict__ p, float * __restrict__ lap, float * __restrict__ coefsx, float * __restrict__ coefsz)
{

   int half_order=order/2;
  	int i =  half_order + blockIdx.x * blockDim.x + threadIdx.x; // Global row index
  	int j =  half_order + blockIdx.y * blockDim.y + threadIdx.y; // Global column index
  	int mult = i*nz;
  	int aux;
	float acmx = 0, acmz = 0;

  	if(i<nx - half_order){
  		if(j<nz - half_order){
			for(int io=0;io<=order;io++){
				aux = io-half_order;
				acmz += p[mult + j+aux]*coefsz[io];
				acmx += p[(i+aux)*nz + j]*coefsx[io];
			}
			lap[mult +j] = acmz + acmx;
			acmx = 0.0;
			acmz = 0.0;
		}
  	}
}

__global__ void kernel_time(int nx, int nz, float *__restrict__ p, float *__restrict__ pp, float *__restrict__ v2, float *__restrict__ lap, float dt2)
{

  	int i =  blockIdx.x * blockDim.x + threadIdx.x; // Global row index
  	int j =  blockIdx.y * blockDim.y + threadIdx.y; // Global column index
  	int mult = i*nz;

  	if(i<nx){
  		if(j<nz){
			 pp[mult+j] = 2.*p[mult+j] - pp[mult+j] + v2[mult+j]*dt2*lap[mult+j];
		}
  	}
}

__global__ void kernel_tapper(int nx, int nz, int nxb, int nzb, float *__restrict__ p, float *__restrict__ pp, float *__restrict__ taperx, float *__restrict__ taperz)
{

	int i =  blockIdx.x * blockDim.x + threadIdx.x; // nx index
	int j =  blockIdx.y * blockDim.y + threadIdx.y; // nzb index
	int itxr = nx - 1, mult = i*nz;

	if(i<nx){
		if(j<nzb){
			p[mult+j] *= taperz[j];
			pp[mult+j] *= taperz[j];
		}
	}

	if(i<nxb){
		if(j<nzb){
			p[mult+j] *= taperx[i];
			pp[mult+j] *= taperx[i];

			p[(itxr-i)*nz+j] *= taperx[i];
			pp[(itxr-i)*nz+j] *= taperx[i];
		}
	}
}

__global__ void kernel_src(int nz, float * __restrict__ pp, int sx, int sz, float srce)
{
 	pp[sx*nz+sz] += srce;
}

__global__ void kernel_upb(int order, int nx, int nz, int nzb, int nt, float *__restrict__ pp, float *__restrict__ upb, int it, int flag)
{
	int half_order = order/2;
	int i = blockIdx.x * blockDim.x + threadIdx.x; //nx index

 	if(i<nx){
		for(int j=nzb-order/2;j<nzb;j++)
    		if(flag == 0)
    			upb[(it*nx*half_order)+(i*half_order)+(j-(nzb-half_order))] = pp[i*nz+j];
        	else
	        	pp[i*nz+j] = upb[((nt-1-it)*nx*half_order)+(i*half_order)+(j-(nzb-half_order))];
  	}
}

__global__ void kernel_sism(int nx, int nz, int nxb, int nt, int is, int it, int gz, float *__restrict__ d_obs, float *__restrict__ ppr)
{
 	int size = nx-(2*nxb);
	int i = blockIdx.x * blockDim.x + threadIdx.x; //nx index
 	if(i<size)
 		ppr[((i+nxb)*nz) + gz] += d_obs[i*nt + (nt-1-it)];

}

__global__ void kernel_img(int nx, int nz, int nxb, int nzb, float * __restrict__ imloc, float * __restrict__ p, float * __restrict__ ppr)
{
 	int size_x = nx-(2*nxb);
 	int size_z = nz-(2*nzb);
	int i =  blockIdx.x * blockDim.x + threadIdx.x; // Global row index
  	int j =  blockIdx.y * blockDim.y + threadIdx.y; // Global column index
 	if(j<size_z){
      if(i<size_x){
        imloc[i*size_z+j] += p[(i+nxb)*nz+(j+nzb)] * ppr[(i+nxb)*nz+(j+nzb)];
      }
    }
}
// ============================ Aux ============================
float *calc_coefs(int order)
{
        float *coef;

        coef = (float *)calloc(order+1,sizeof(float));
        switch(order)
        {
                case 2:
                        coef[0] = 1.;
                        coef[1] = -2.;
                        coef[2] = 1.;
                        break;
                case 4:
                        coef[0] = -1./12.;
                        coef[1] = 4./3.;
                        coef[2] = -5./2.;
                        coef[3] = 4./3.;
                        coef[4] = -1./12.;
                        break;
                case 6:
                        coef[0] = 1./90.;
                        coef[1] = -3./20.;
                        coef[2] = 3./2.;
                        coef[3] = -49./18.;
                        coef[4] = 3./2.;
                        coef[5] = -3./20.;
                        coef[6] = 1./90.;
                        break;
                case 8:

                        coef[0] = -1./560.;
                        coef[1] = 8./315.;
                        coef[2] = -1./5.;
                        coef[3] = 8./5.;
                        coef[4] = -205./72.;
                        coef[5] = 8./5.;
                        coef[6] = -1./5.;
                        coef[7] = 8./315.;
                        coef[8] = -1./560.;
                        break;
                default:
                        makeo2(coef,order);
        }

        return coef;
}

static void makeo2 (float *coef,int order)
{
        float h_beta, alpha1=0.0;
        float alpha2=0.0;
        float  central_term=0.0;
        float coef_filt=0;
        float arg=0.0;
        float  coef_wind=0.0;
        int msign,ix;

        float alpha = .54;
        float beta = 6.;
        h_beta = 0.5*beta;
        alpha1=2.*alpha-1.0;
        alpha2=2.*(1.0-alpha);
        central_term=0.0;

        msign=-1;

        for (ix=1; ix <= order/2; ix++){
                msign=-msign ;
                coef_filt = (2.*msign)/(ix*ix);
                arg = PI*ix/(2.*(order/2+2));
                coef_wind=pow((alpha1+alpha2*cos(arg)*cos(arg)),h_beta);
                coef[order/2+ix] = coef_filt*coef_wind;
                central_term = central_term + coef[order/2+ix];
                coef[order/2-ix] = coef[order/2+ix];
        }

        coef[order/2]  = -2.*central_term;

        return;
}

void *alloc1 (size_t n1, size_t size)
{
	void *p;

	if ((p=malloc(n1*size))==NULL)
		return NULL;
	return p;
}

void **alloc2 (size_t n1, size_t n2, size_t size)
{
	size_t i2;
	void **p;

	if ((p=(void**)malloc(n2*sizeof(void*)))==NULL) 
		return NULL;
	if ((p[0]=(void*)malloc(n2*n1*size))==NULL) {
		free(p);
		return NULL;
	}
	for (i2=0; i2<n2; i2++)
		p[i2] = (char*)p[0]+size*n1*i2;
	return p;
}

void ***alloc3 (size_t n1, size_t n2, size_t n3, size_t size)
{
	size_t i3,i2;
	void ***p;

	if ((p=(void***)malloc(n3*sizeof(void**)))==NULL)
		return NULL;
	if ((p[0]=(void**)malloc(n3*n2*sizeof(void*)))==NULL) {
		free(p);
		return NULL;
	}
	if ((p[0][0]=(void*)malloc(n3*n2*n1*size))==NULL) {
		free(p[0]);
		free(p);
		return NULL;
	}

	for (i3=0; i3<n3; i3++) {
		p[i3] = p[0]+n2*i3;
		for (i2=0; i2<n2; i2++)
			p[i3][i2] = (char*)p[0][0]+size*n1*(i2+n2*i3);
	}
	return p;
}

float *alloc1float(size_t n1)
{
	return (float*)alloc1(n1,sizeof(float));
}

float **alloc2float(size_t n1, size_t n2)
{
	return (float**)alloc2(n1,n2,sizeof(float));
}

float ***alloc3float(size_t n1, size_t n2, size_t n3)
{
	return (float***)alloc3(n1,n2,n3,sizeof(float));
}

void free1 (void *p)
{
	free(p);
}

void free2 (void **p)
{
	free(p[0]);
	free(p);
}

void free3 (void ***p)
{
	free(p[0][0]);
	free(p[0]);
	free(p);
}

void free1float(float *p)
{
	free1(p);
}

void free2float(float **p)
{
	free2((void**)p);
}

void free3float(float ***p)
{
	free3((void***)p);
}
// ============================ Init ============================
void fd_init_cuda(int order, int nxe, int nze, int nxb, int nzb, int nt, int ns, float fac)
{
        float dfrac;
	// cudaProfilerSart();
   	nxbin=nxb; nzbin=nzb;
   	brdBufferLength = nxb*sizeof(float);
   	mtxBufferLength = (nxe*nze)*sizeof(float);
   	coefsBufferLength = (order+1)*sizeof(float);
   	upbBufferLength = nt*nxe*(order/2)*sizeof(float);
	obsBufferLength = nt*(nxe-(2*nxb))*sizeof(float);
   	imgBufferLength = (nxe-(2*nxb))*(nze-(2*nzb))*sizeof(float);

	taper_x = alloc1float(nxb);
	taper_z = alloc1float(nzb);

	dfrac = sqrt(-log(fac))/(1.*nxb);
	for(int i=0;i<nxb;i++)
	  taper_x[i] = exp(-pow((dfrac*(nxb-i)),2));


	dfrac = sqrt(-log(fac))/(1.*nzb);
	for(int i=0;i<nzb;i++)
	  taper_z[i] = exp(-pow((dfrac*(nzb-i)),2));


	// Create a Device pointers
	hipMalloc((void **) &d_v2, mtxBufferLength);
	hipMalloc((void **) &d_p, mtxBufferLength);
	hipMalloc((void **) &d_pp, mtxBufferLength);
	hipMalloc((void **) &d_pr, mtxBufferLength);
	hipMalloc((void **) &d_ppr, mtxBufferLength);
	hipMalloc((void **) &d_swap, mtxBufferLength);
	hipMalloc((void **) &d_laplace, mtxBufferLength);

	hipMalloc((void **) &d_upb, upbBufferLength);
	hipMalloc((void **) &d_sis, obsBufferLength);
	hipMalloc((void **) &d_img, imgBufferLength);
	hipMalloc((void **) &d_coefs_x, coefsBufferLength);
	hipMalloc((void **) &d_coefs_z, coefsBufferLength);
	hipMalloc((void **) &d_taperx, brdBufferLength);
	hipMalloc((void **) &d_taperz, brdBufferLength);

	int div_x, div_z;
	// Set a Grid for the execution on the device
	div_x = (float) nxe/(float) sizeblock;
	div_z = (float) nze/(float) sizeblock;
	gridx = (int) ceil(div_x);
	gridz = (int) ceil(div_z);

	div_x = (float) nxb/(float) sizeblock;
	div_z = (float) nzb/(float) sizeblock;
	gridBorder_x = (int) ceil(div_x);
	gridBorder_z = (int) ceil(div_z);

	div_x = (float) 8/(float) sizeblock;
	gridupb = (int) ceil(div_x);
}

void fd_init(int order, int nx, int nz, int nxb, int nzb, int nt, int ns, float fac, float dx, float dz, float dt)
{
        int io;
	dx2inv = (1./dx)*(1./dx);
        dz2inv = (1./dz)*(1./dz);
	dt2 = dt*dt;

	coefs = calc_coefs(order);
	laplace = alloc2float(nz,nx);

	coefs_z = calc_coefs(order);
	coefs_x = calc_coefs(order);

	// pre calc coefs 8 d2 inv
	for (io = 0; io <= order; io++) {
		coefs_z[io] = dz2inv * coefs[io];
		coefs_x[io] = dx2inv * coefs[io];
	}

	memset(*laplace,0,nz*nx*sizeof(float));

        fd_init_cuda(order,nx,nz,nxb,nzb,nt,ns,fac);

        return;
}

void write_buffers(float **p, float **pp, float **v2, float ***upb, float *taperx, float *taperz, float **d_obs, float **imloc, int is, int flag)
{
    
        if(flag == 0){
                hipMemcpy(d_p, p[0], mtxBufferLength, hipMemcpyHostToDevice);
                hipMemcpy(d_pp, pp[0], mtxBufferLength, hipMemcpyHostToDevice);
                hipMemcpy(d_v2, v2[0], mtxBufferLength, hipMemcpyHostToDevice);
                hipMemcpy(d_coefs_x, coefs_x, coefsBufferLength, hipMemcpyHostToDevice);
                hipMemcpy(d_coefs_z, coefs_z, coefsBufferLength, hipMemcpyHostToDevice);
                hipMemcpy(d_taperx, taperx, brdBufferLength, hipMemcpyHostToDevice);
                hipMemcpy(d_taperz, taperz, brdBufferLength, hipMemcpyHostToDevice);
                hipMemcpy(d_upb, upb[0][0], upbBufferLength, hipMemcpyHostToDevice);
        }

        if(flag == 1){
                hipMemcpy(d_pr, p[0], mtxBufferLength, hipMemcpyHostToDevice);
                hipMemcpy(d_ppr, pp[0], mtxBufferLength, hipMemcpyHostToDevice);
                hipMemcpy(d_sis, d_obs[is], obsBufferLength, hipMemcpyHostToDevice);
                hipMemcpy(d_img, imloc[0], imgBufferLength, hipMemcpyHostToDevice);
        }
}
// ============================ Propagation ============================
void fd_forward(int order, float **p, float **pp, float **v2, float ***upb, int nz, int nx, int nt, int is, int sz, int *sx, float *srce, int propag)
{
 	dim3 dimGrid(gridx, gridz);
  	dim3 dimGridTaper(gridx, gridBorder_z);

  	dim3 dimGridSingle(1,1);
  	dim3 dimGridUpb(gridx,1);

  	dim3 dimBlock(sizeblock, sizeblock);
  	
	write_buffers(p,pp,v2,upb,taper_x, taper_z,NULL, NULL,is,0);
	   	
   	for (int it = 0; it < nt; it++){
	 	d_swap  = d_pp;
	 	d_pp = d_p;
	 	d_p = d_swap;

	 	kernel_tapper<<<dimGridTaper, dimBlock>>>(nx,nz,nxbin,nzbin,d_p,d_pp,d_taperx,d_taperz);
	 	kernel_lap<<<dimGrid, dimBlock>>>(order,nx,nz,d_p,d_laplace,d_coefs_x,d_coefs_z);
	 	kernel_time<<<dimGrid, dimBlock>>>(nx,nz,d_p,d_pp,d_v2,d_laplace,dt2);
	 	kernel_src<<<dimGridSingle, dimBlock>>>(nz,d_pp,sx[is],sz,srce[it]);
	 	kernel_upb<<<dimGridUpb, dimBlock>>>(order,nx,nz,nzbin,nt,d_pp,d_upb,it,0);
		cudaCheck();

     	        if((it+1)%100 == 0){fprintf(stdout,"\r* it = %d / %d (%d%)",it+1,nt,(100*(it+1)/nt));fflush(stdout);}
 	}
 	hipMemcpy(p[0], d_p, mtxBufferLength, hipMemcpyDeviceToHost);
 	hipMemcpy(pp[0], d_pp, mtxBufferLength, hipMemcpyDeviceToHost);
 	hipMemcpy(upb[0][0], d_upb, upbBufferLength, hipMemcpyDeviceToHost);
}

void fd_back(int order, float **p, float **pp, float **pr, float **ppr, float **v2, float ***upb, int nz, int nx, int nt, int is, int sz, int gz, float ***snaps, float **imloc, float **d_obs)
{
	int ix, iz, it;

	dim3 dimGrid(gridx, gridz);
  	dim3 dimGridTaper(gridx, gridBorder_z);
  	dim3 dimGridUpb(gridx,1);

  	dim3 dimBlock(sizeblock, sizeblock);
	write_buffers(p,pp,v2,upb,taper_x, taper_z,d_obs,imloc,is,0);
	write_buffers(pr,ppr,v2,upb,taper_x,taper_z,d_obs,imloc,is,1);
	
        for(it=0; it<nt; it++)
        {
                if(it==0 || it==1)
                {
                        for(ix=0; ix<nx; ix++)
                        {
                                for(iz=0; iz<nz; iz++)
                                {
                                        pp[ix][iz] = snaps[1-it][ix][iz];
                                }
                        }
                        hipMemcpy(d_pp, pp[0], mtxBufferLength, hipMemcpyHostToDevice);
                }
                else
                {
                        kernel_lap<<<dimGrid, dimBlock>>>(order,nx,nz,d_p,d_laplace,d_coefs_x,d_coefs_z);
                        kernel_time<<<dimGrid, dimBlock>>>(nx,nz,d_p,d_pp,d_v2,d_laplace,dt2);
                        kernel_upb<<<dimGridUpb, dimBlock>>>(order,nx,nz,nzbin,nt,d_pp,d_upb,it,1);
                }

                d_swap = d_pp;
                d_pp = d_p;
                d_p = d_swap;

                kernel_tapper<<<dimGridTaper, dimBlock>>>(nx,nz,nxbin,nzbin,d_pr,d_ppr,d_taperx,d_taperz);
                kernel_lap<<<dimGrid, dimBlock>>>(order,nx,nz,d_pr,d_laplace, d_coefs_x, d_coefs_z);
                kernel_time<<<dimGrid, dimBlock>>>(nx,nz,d_pr,d_ppr,d_v2,d_laplace,dt2);
                kernel_sism<<<dimGridUpb, dimBlock>>>(nx,nz,nxbin,nt,is,it,gz,d_sis,d_ppr);
                kernel_img<<<dimGrid, dimBlock>>>(nx,nz,nxbin,nzbin,d_img,d_p,d_ppr);

                d_swap = d_ppr;
                d_ppr = d_pr;
                d_pr = d_swap;

                if((it+1)%100 == 0)
                {
                        fprintf(stdout,"\r* it = %d / %d (%d%)",it+1,nt,(100*(it+1)/nt));fflush(stdout);
                }
	}
}

int main (int argc, char **argv)
{
	FILE *fsource = NULL, *fvel_ext = NULL, *fd_obs = NULL, *fvp = NULL, *fsns = NULL,*fsns2 = NULL, *fsnr = NULL, *fimg = NULL, *flim = NULL, *fimg_lap = NULL;

	int iz, ix, it, is;

	float *srce;
	float **vp = NULL, **vpe = NULL, **vpex = NULL;

	float **PP,**P,**PPR,**PR,**tmp;
	float ***swf, ***upb, ***snaps, **vel2, ***d_obs, ***vel_ext_rnd;
	float **imloc, **img, **img_lap;
        read_input(argv[1]);

        printf("## vp = %s, d_obs = %s, vel_ext_file = %s, vel_ext_flag = %d \n",vpfile,datfile,vel_ext_file,vel_ext_flag);
	printf("## nz = %d, nx = %d, nt = %d \n",nz,nx,nt);
	printf("## dz = %f, dx = %f, dt = %f \n",dz,dx,dt);
	printf("## ns = %d, sz = %d, fsx = %d, ds = %d, gz = %d \n",ns,sz,fsx,ds,gz);
	printf("## order = %d, nzb = %d, nxb = %d, F = %f, rnd = %d \n",order,nzb,nxb,fac,rnd);
        srce = alloc1float(nt);
        //ricker_wavelet(nt, dt, fpeak, srce);
	sx = alloc1int(ns);
	for(is=0; is<ns; is++){
		sx[is] = fsx + is*ds + nxb;
	}
	sz += nzb;
	gz += nzb;
	nze = nz + 2 * nzb;
	nxe = nx + 2 * nxb;
	if(vel_ext_flag){
		vel_ext_rnd = alloc3float(nze,nxe,ns);
		memset(**vel_ext_rnd,0,nze*nxe*ns*sizeof(float));
		fvel_ext = fopen(vel_ext_file,"r");
		fread(**vel_ext_rnd,sizeof(float),nze*nxe*ns,fvel_ext);
		fclose(fvel_ext);
	}

	d_obs = alloc3float(nt,nx,ns);
	memset(**d_obs,0,nt*nx*ns*sizeof(float));
	fd_obs = fopen(datfile,"r");
	fread(**d_obs,sizeof(float),nt*nx*ns,fd_obs);
	fclose(fd_obs);

	float **d_obs_aux=(float**)malloc(ns*sizeof(float*));
	for(int i=0; i<ns; i++) 
		d_obs_aux[i] = (float*)malloc((nt*nx)*sizeof(float)); 
	
	for(int i=0; i<ns; i++){
		for(int j=0; j<nx; j++){
			for(int k=0; k<nt; k++)
				d_obs_aux[i][j*nt+k] = d_obs[i][j][k]; 
		}
	}

	vp = alloc2float(nz,nx);
	memset(*vp,0,nz*nx*sizeof(float));
	fvp = fopen(vpfile,"r");
	fread(vp[0],sizeof(float),nz*nx,fvp);
	fclose(fvp);

	vpe = alloc2float(nze,nxe);
	vpex = vpe;

	for(ix=0; ix<nx; ix++){
		for(iz=0; iz<nz; iz++){
			vpe[ix+nxb][iz+nzb] = vp[ix][iz]; 
		}
	}

	vel2 = alloc2float(nze,nxe);
        fd_init(order,nxe,nze,nxb,nzb,nt,ns,fac,dx,dz,dt);
	//taper_init(nxb,nzb,fac);

        PP = alloc2float(nze,nxe);
	P = alloc2float(nze,nxe);
	PPR = alloc2float(nze,nxe);
	PR = alloc2float(nze,nxe);
	upb = alloc3float(order/2,nxe,nt);
	snaps = alloc3float(nze,nxe,2);
	imloc = alloc2float(nz,nx);
	img = alloc2float(nz,nx);
	img_lap = alloc2float(nz,nx);

	char filepath [100];
	sprintf(filepath, "%s/dir.snaps", tmpdir);
	fsns = fopen(filepath,"w");
	sprintf(filepath, "%s/dir.snaps_rec", tmpdir);
	fsns2 = fopen(filepath,"w");
	sprintf(filepath, "%s/dir.snapr", tmpdir);
	fsnr = fopen(filepath,"w");
	sprintf(filepath, "%s/dir.image", tmpdir);
	fimg = fopen(filepath,"w");
	sprintf(filepath, "%s/dir.image_lap", tmpdir);	
	fimg_lap = fopen(filepath,"w");
	
	memset(*img,0,nz*nx*sizeof(float));
        memset(*img_lap,0,nz*nx*sizeof(float));
        
        for(is=0; is<ns; is++){
		fprintf(stdout,"** source %d, at (%d,%d) \n",is+1,sx[is]-nxb,sz-nzb);

		if (vel_ext_flag){
			vpe = vel_ext_rnd[is];					// load hybrid border vpe from file
		}else{
			extendvel_linear(nx,nz,nxb,nzb,vpe); 	// hybrid border (linear randomic)
		}


		for(ix=0; ix<nx+2*nxb; ix++){
			for(iz=0; iz<nz+2*nzb; iz++){
				vel2[ix][iz] = vpe[ix][iz]*vpe[ix][iz];
			}
		}

		memset(*PP,0,nze*nxe*sizeof(float));
		memset(*P,0,nze*nxe*sizeof(float));
    
		fd_forward(order,P,PP,vel2,upb,nze,nxe,nt,is,sz,sx,srce, is);
		fprintf(stdout,"\n");

		for(iz=0; iz<nze; iz++){
			for(ix=0; ix<nxe; ix++){
				snaps[0][ix][iz] = P[ix][iz];
				snaps[1][ix][iz] = PP[ix][iz];
			}
		}

		fprintf(stdout,"** backward propagation %d, at (%d,%d) \n",is+1,sx[is]-nxb,sz-nzb);

		memset(*PP,0,nze*nxe*sizeof(float));
		memset(*P,0,nze*nxe*sizeof(float));
		memset(*PPR,0,nze*nxe*sizeof(float));
		memset(*PR,0,nze*nxe*sizeof(float));
		memset(*imloc,0,nz*nx*sizeof(float));


		fd_back(order,P,PP,PR,PPR,vel2,upb,nze,nxe,nt,is,sz,gz,snaps,imloc,d_obs_aux);
		fprintf(stdout,"\n");
		

		for(iz=0; iz<nz; iz++){
			for(ix=0; ix<nx; ix++){
				img[ix][iz] += imloc[ix][iz];
			}
		}
	}
	fwrite(*img,sizeof(float),nz*nx,fimg);

	fwrite(*img_lap,sizeof(float),nz*nx,fimg_lap);

	fclose(fsns);
	fclose(fsns2);
	fclose(fsnr);
	fclose(fimg);
	fclose(fimg_lap);
        
        // free memory device
        // taper_destroy();
        free1float(coefs);
	free1int(sx);
	free1float(srce);
        free2float(laplace);
	free2float(vp);
	free2float(P);
	free2float(PP);
	free2float(PR);
	free2float(PPR);
	free3float(snaps);
	free2float(imloc);
	free2float(img);
	free2float(img_lap);
	free2float(vpex);
	free2float(vel2);
	free3float(upb);
        free3float(d_obs);
        if(vel_ext_flag) free3float(vel_ext_rnd);
        hipFree(d_p);
        hipFree(d_pp);
        hipFree(d_pr);
        hipFree(d_ppr);
        hipFree(d_v2);
        hipFree(d_laplace);
        hipFree(d_coefs_z);
        hipFree(d_coefs_x);

        hipFree(d_taperx);
        hipFree(d_taperz);

        hipFree(d_sis);
        hipFree(d_img);
        hipFree(d_upb);
        return 0;
}
