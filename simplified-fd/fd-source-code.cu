#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

void fd_init(int order, int nx, int nz, float dx, float dz);
void fd_init_cuda(int order, int nxe, int nze);
float *calc_coefs(int order);
static void makeo2 (float *coef,int order);

#define sizeblock 32
#define PI (3.141592653589793)

float *d_p;
float *d_laplace, *d_coefs_x, *d_coefs_z;

size_t mtxBufferLength, coefsBufferLength;

int gridx, gridz;

static float dx2inv, dz2inv;
static float *coefs = NULL;
static float *coefs_z = NULL;
static float *coefs_x = NULL;

__global__ void kernel_lap(int order, int nx, int nz, float * __restrict__ p, float * __restrict__ lap, float * __restrict__ coefsx, float * __restrict__ coefsz)
{
        int half_order=order/2;
        int i =  half_order + blockIdx.x * blockDim.x + threadIdx.x; // Global row index
        int j =  half_order + blockIdx.y * blockDim.y + threadIdx.y; // Global column index
        int mult = i*nz;
        int aux;
        float acmx = 0, acmz = 0;

        if(i<nx - half_order)
        {
                if(j<nz - half_order)
                {
                        for(int io=0;io<=order;io++)
                        {
                                aux = io-half_order;
                                acmz += p[mult + j+aux]*coefsz[io];
                                acmx += p[(i+aux)*nz + j]*coefsx[io];
                        }
                        lap[mult +j] = acmz + acmx;
                        acmx = 0.0;
                        acmz = 0.0;
                }
        }

}

float *calc_coefs(int order)
{
        float *coef;

        coef = (float *)calloc(order+1,sizeof(float));
        switch(order)
        {
                case 2:
                        coef[0] = 1.;
                        coef[1] = -2.;
                        coef[2] = 1.;
                        break;
                case 4:
                        coef[0] = -1./12.;
                        coef[1] = 4./3.;
                        coef[2] = -5./2.;
                        coef[3] = 4./3.;
                        coef[4] = -1./12.;
                        break;
                case 6:
                        coef[0] = 1./90.;
                        coef[1] = -3./20.;
                        coef[2] = 3./2.;
                        coef[3] = -49./18.;
                        coef[4] = 3./2.;
                        coef[5] = -3./20.;
                        coef[6] = 1./90.;
                        break;
                case 8:

                        coef[0] = -1./560.;
                        coef[1] = 8./315.;
                        coef[2] = -1./5.;
                        coef[3] = 8./5.;
                        coef[4] = -205./72.;
                        coef[5] = 8./5.;
                        coef[6] = -1./5.;
                        coef[7] = 8./315.;
                        coef[8] = -1./560.;
                        break;
                default:
                        makeo2(coef,order);
        }

        return coef;
}

static void makeo2 (float *coef,int order){
        float h_beta, alpha1=0.0;
        float alpha2=0.0;
        float  central_term=0.0;
        float coef_filt=0;
        float arg=0.0;
        float  coef_wind=0.0;
        int msign,ix;

        float alpha = .54;
        float beta = 6.;
        h_beta = 0.5*beta;
        alpha1=2.*alpha-1.0;
        alpha2=2.*(1.0-alpha);
        central_term=0.0;

        msign=-1;

        for (ix=1; ix <= order/2; ix++){
                msign=-msign ;
                coef_filt = (2.*msign)/(ix*ix);
                arg = PI*ix/(2.*(order/2+2));
                coef_wind=pow((alpha1+alpha2*cos(arg)*cos(arg)),h_beta);
                coef[order/2+ix] = coef_filt*coef_wind;
                central_term = central_term + coef[order/2+ix];
                coef[order/2-ix] = coef[order/2+ix];
        }

        coef[order/2]  = -2.*central_term;

        return;
}

void fd_init_cuda(int order, int nxe, int nze)
{
        mtxBufferLength = (nxe*nze)*sizeof(float);
        coefsBufferLength = (order+1)*sizeof(float);

        // Create a Device pointers
        hipMalloc(&d_p, mtxBufferLength);
        hipMalloc(&d_laplace, mtxBufferLength);
        hipMalloc(&d_coefs_x, coefsBufferLength);
        hipMalloc(&d_coefs_z, coefsBufferLength);

        int div_x, div_z;
        // Set a Grid for the execution on the device
        int tx = ((nxe - 1) / 32 + 1) * 32;
        int tz = ((nze - 1) / 32 + 1) * 32;

        div_x = (float) tx/(float) sizeblock;
        div_z = (float) tz/(float) sizeblock;

        gridx = (int) ceil(div_x);
        gridz = (int) ceil(div_z);
}

void fd_init(int order, int nx, int nz, float dx, float dz)
{
        int io;
        dx2inv = (1./dx)*(1./dx);
        dz2inv = (1./dz)*(1./dz);

        coefs = calc_coefs(order);

        coefs_z = calc_coefs(order);
        coefs_x = calc_coefs(order);

        // pre calc coefs 8 d2 inv
        for (io = 0; io <= order; io++)
        {
                coefs_z[io] = dz2inv * coefs[io];
                coefs_x[io] = dx2inv * coefs[io];
        }

        fd_init_cuda(order,nx,nz);

        return;
}

int main (int argc, char **argv)
{
        // constantes
        int nz = 195, nx = 315, nxb = 50, nzb = 50, nxe, nze, order = 8;
        float dz = 10.000000, dx = 10.000000;

        nxe = nx + 2 * nxb;
        nze = nz + 2 * nzb;
        // inicialização
        fd_init(order,nxe,nze,dx,dz);
        
        dim3 dimGrid(gridx, gridz);
        dim3 dimBlock(sizeblock, sizeblock);

        
        FILE *finput;
        // leitura do input
        finput = fopen("input.bin", "rb");

        float input_data[mtxBufferLength];
        printf("lendo arquivo...\n");
        fread(input_data, sizeof(input_data), 1, finput);
        fclose(finput);

        // utilização do kernel
        hipMemcpy(d_p, input_data, mtxBufferLength, hipMemcpyHostToDevice);
        hipMemcpy(d_coefs_x, coefs_x, coefsBufferLength, hipMemcpyHostToDevice);
        hipMemcpy(d_coefs_z, coefs_z, coefsBufferLength, hipMemcpyHostToDevice);

        kernel_lap<<<dimGrid, dimBlock>>>(order,nx,nz,d_p,d_laplace,d_coefs_x,d_coefs_z);

        float output_data[mtxBufferLength];
        
        hipMemcpy(output_data, d_laplace, mtxBufferLength, hipMemcpyDeviceToHost);
        
        // salvando a saída
        FILE *foutput;
        printf("salvando saída...\n");
        foutput = fopen("output_cuda.bin", "wb");
        fwrite(output_data, sizeof(output_data), 1, foutput);
        fclose(foutput);

        // free memory device

        hipFree(d_p);
        hipFree(d_laplace);
        hipFree(d_coefs_x);
        hipFree(d_coefs_z);
        return 0;
}
