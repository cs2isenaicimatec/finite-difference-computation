#include <stdio.h>
#include <hip/hip_runtime.h>

void fd_init(int order, int nx, int nz, float dx, float dz);
void fd_init_cuda(int order, int nxe, int nze);
float *calc_coefs(int order);

#define sizeblock 32

float *d_p;
float *d_laplace, *d_coefs_x, *d_coefs_z;

size_t mtxBufferLength, coefsBufferLength;

int gridx, gridz;

static float dx2inv, dz2inv;
static float *coefs = NULL;
static float *coefs_z = NULL;
static float *coefs_x = NULL;

__global__ void kernel_lap(int order, int nx, int nz, float * __restrict__ p, float * __restrict__ lap, float * __restrict__ coefsx, float * __restrict__ coefsz)
{
        int half_order=order/2;
        int i =  half_order + blockIdx.x * blockDim.x + threadIdx.x; // Global row index
        int j =  half_order + blockIdx.y * blockDim.y + threadIdx.y; // Global column index
        int mult = i*nz;
        int aux;
        float acmx = 0, acmz = 0;

        if(i<nx - half_order)
        {
                if(j<nz - half_order)
                {
                        for(int io=0;io<=order;io++)
                        {
                                aux = io-half_order;
                                acmz += p[mult + j+aux]*coefsz[io];
                                acmx += p[(i+aux)*nz + j]*coefsx[io];
                        }
                        lap[mult +j] = acmz + acmx;
                        acmx = 0.0;
                        acmz = 0.0;
                }
        }

}

float *calc_coefs(int order)
{
        float *coef;

        coef = (float *)calloc(order+1,sizeof(float));
        switch(order)
        {
                case 2:
                        coef[0] = 1.;
                        coef[1] = -2.;
                        coef[2] = 1.;
                        break;
                case 4:
                        coef[0] = -1./12.;
                        coef[1] = 4./3.;
                        coef[2] = -5./2.;
                        coef[3] = 4./3.;
                        coef[4] = -1./12.;
                        break;
                case 6:
                        coef[0] = 1./90.;
                        coef[1] = -3./20.;
                        coef[2] = 3./2.;
                        coef[3] = -49./18.;
                        coef[4] = 3./2.;
                        coef[5] = -3./20.;
                        coef[6] = 1./90.;
                        break;
                case 8:
                        coef[0] = -1./560.;
                        coef[1] = 8./315.;
                        coef[2] = -1./5.;
                        coef[3] = 8./5.;
                        coef[4] = -205./72.;
                        coef[5] = 8./5.;
                        coef[6] = -1./5.;
                        coef[7] = 8./315.;
                        coef[8] = -1./560.;
                        break;
        }

        return coef;
}

void fd_init_cuda(int order, int nxe, int nze)
{
        mtxBufferLength = (nxe*nze)*sizeof(float);
        coefsBufferLength = (order+1)*sizeof(float);

        // Create a Device pointers
        hipMalloc(&d_p, mtxBufferLength);
        hipMalloc(&d_laplace, mtxBufferLength);
        hipMalloc(&d_coefs_x, coefsBufferLength);
        hipMalloc(&d_coefs_z, coefsBufferLength);

        int div_x, div_z;
        // Set a Grid for the execution on the device
        int tx = ((nxe - 1) / 32 + 1) * 32;
        int tz = ((nze - 1) / 32 + 1) * 32;



        div_x = (float) tx/(float) sizeblock;
        div_z = (float) tz/(float) sizeblock;



        gridx = (int) ceil(div_x);
        gridz = (int) ceil(div_z);
}

void fd_init(int order, int nx, int nz, float dx, float dz)
{
        int io;
        dx2inv = (1./dx)*(1./dx);
        dz2inv = (1./dz)*(1./dz);

        coefs = calc_coefs(order);

        coefs_z = calc_coefs(order);
        coefs_x = calc_coefs(order);

        // pre calc coefs 8 d2 inv
        for (io = 0; io <= order; io++)
        {
                coefs_z[io] = dz2inv * coefs[io];
                coefs_x[io] = dx2inv * coefs[io];
        }


        fd_init_cuda(order,nx,nz);

        return;
}

int main (int argc, char **argv)
{
        // constantes
        int nz = 195, nx = 315, nxb = 50, nzb = 50, nxe, nze, order = 8;
        float dz = 10.00000, dx = 10.000000;



        nxe = nx + 2 * nxb;
        nze = nz + 2 * nzb;
        // inicialização
        fd_init(order,nxe,nze,dx,dz);
        dim3 dimGrid(gridx, gridz);
        dim3 dimBlock(sizeblock, sizeblock);


        // arquivos
        FILE *finput;
        FILE *foutput;
        // leitura do input
        finput = fopen("./input.bin", "rb");

        float input_data[mtxBufferLength], output_data[mtxBufferLength];
        printf("lendo arquivo...\n");
        fread(input_data, sizeof(input_data), 1, finput);
        printf("%.15f\n", input_data[1341]);
        fclose(finput);
        // utilização do kernel
        hipMemcpy(d_p, input_data, mtxBufferLength, hipMemcpyHostToDevice);


        kernel_lap<<<dimGrid, dimBlock>>>(order,nx,nz,d_p,d_laplace,d_coefs_x,d_coefs_z);

        hipMemcpy(output_data, d_laplace, mtxBufferLength, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if(error != hipSuccess)
        {
                // print the CUDA error message and exit
                printf("CUDA error: %s\n", hipGetErrorString(error));
                exit(-1);
        }

        // salvando a saída
        printf("salvando saída...\n");
        foutput = fopen("output_teste.bin", "wb");
        printf("%.15f\n", output_data[1341]);
        printf("Esperado: 0.000010451854905\n");
        printf("escrevendo arquivo\n");
        fwrite(output_data, sizeof(output_data), 1, foutput);
        fclose(foutput);
        // free memory device

        hipFree(d_p);
        hipFree(d_laplace);
        hipFree(d_coefs_x);
        hipFree(d_coefs_z);
        return 0;
}
