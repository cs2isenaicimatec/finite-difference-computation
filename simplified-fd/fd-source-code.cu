#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>

void fd_init(int order, int nx, int nz, float dx, float dz);
void fd_init_cuda(int order, int nxe, int nze);
float *calc_coefs(int order);
static void makeo2 (float *coef,int order);
void read_input(char *file);

#define sizeblock 32
#define PI (3.141592653589793)

char *file_path;

float *d_p;
float *d_laplace, *d_coefs_x, *d_coefs_z;

size_t mtxBufferLength, coefsBufferLength;

int gridx, gridz;
int nz, nx, nxb, nzb, nxe, nze, order;
float dz, dx;

static float dx2inv, dz2inv;
static float *coefs = NULL;
static float *coefs_z = NULL;
static float *coefs_x = NULL;

void read_input(char *file)
{
        FILE *fp;
        fp = fopen(file, "r");
        char *line = NULL;
        size_t len = 0;
        if (fp == NULL)
                exit(EXIT_FAILURE);
        while (getline(&line, &len, fp) != -1) {
                if(strstr(line,"tmpdir") != NULL)
                {
                        char *tok;
                        tok = strtok(line, "=");
                        tok = strtok(NULL,"=");
                        tok[strlen(tok) - 2] = '\0';
                        file_path = strdup(tok);
                }
                if(strstr(line,"nzb") != NULL)
                {
                        char *nzb_char;
                        nzb_char = strtok(line, "=");
                        nzb_char = strtok(NULL,"=");
                        nzb = atoi(nzb_char);
                }
                if(strstr(line,"nxb") != NULL)
                {
                        char *nxb_char;
                        nxb_char = strtok(line, "=");
                        nxb_char = strtok(NULL,"=");
                        nxb = atoi(nxb_char);
                }
                if(strstr(line,"nz") != NULL)
                {
                        char *nz_char;
                        nz_char = strtok(line, "=");
                        if (strlen(nz_char) <= 2)
                        {
                                nz_char = strtok(NULL,"=");
                                nz = atoi(nz_char);
                        }
                }
                if(strstr(line,"nx") != NULL)
                {
                        char *nx_char;
                        nx_char = strtok(line, "=");
                        if (strlen(nx_char) <= 2)
                        {
                                nx_char = strtok(NULL,"=");
                                nx = atoi(nx_char);
                        }
                }
                if(strstr(line,"dz") != NULL)
                {
                        char *dz_char;
                        dz_char = strtok(line, "=");
                        dz_char = strtok(NULL,"=");
                        dz = atof(dz_char);
                }
                if(strstr(line,"dx") != NULL)
                {
                        char *dx_char;
                        dx_char = strtok(line, "=");
                        dx_char = strtok(NULL,"=");
                        dx = atof(dx_char);
                }
                if(strstr(line,"order") != NULL)
                {
                        char *order_char;
                        order_char = strtok(line, "=");
                        order_char = strtok(NULL,"=");
                        order = atoi(order_char);
                }
        }
				free(line);
}

__global__ void kernel_lap(int order, int nx, int nz, float * __restrict__ p, float * __restrict__ lap, float * __restrict__ coefsx, float * __restrict__ coefsz)
{
        int half_order=order/2;
        int i =  half_order + blockIdx.x * blockDim.x + threadIdx.x; // Global row index
        int j =  half_order + blockIdx.y * blockDim.y + threadIdx.y; // Global column index
        int mult = i*nz;
        int aux;
        float acmx = 0, acmz = 0;

        if(i<nx - half_order)
        {
                if(j<nz - half_order)
                {
                        for(int io=0;io<=order;io++)
                        {
                                aux = io-half_order;
                                acmz += p[mult + j+aux]*coefsz[io];
                                acmx += p[(i+aux)*nz + j]*coefsx[io];
                        }
                        lap[mult +j] = acmz + acmx;
                        acmx = 0.0;
                        acmz = 0.0;
                }
        }

}

float *calc_coefs(int order)
{
        float *coef;

        coef = (float *)calloc(order+1,sizeof(float));
        switch(order)
        {
                case 2:
                        coef[0] = 1.;
                        coef[1] = -2.;
                        coef[2] = 1.;
                        break;
                case 4:
                        coef[0] = -1./12.;
                        coef[1] = 4./3.;
                        coef[2] = -5./2.;
                        coef[3] = 4./3.;
                        coef[4] = -1./12.;
                        break;
                case 6:
                        coef[0] = 1./90.;
                        coef[1] = -3./20.;
                        coef[2] = 3./2.;
                        coef[3] = -49./18.;
                        coef[4] = 3./2.;
                        coef[5] = -3./20.;
                        coef[6] = 1./90.;
                        break;
                case 8:

                        coef[0] = -1./560.;
                        coef[1] = 8./315.;
                        coef[2] = -1./5.;
                        coef[3] = 8./5.;
                        coef[4] = -205./72.;
                        coef[5] = 8./5.;
                        coef[6] = -1./5.;
                        coef[7] = 8./315.;
                        coef[8] = -1./560.;
                        break;
                default:
                        makeo2(coef,order);
        }

        return coef;
}

static void makeo2 (float *coef,int order)
{
        float h_beta, alpha1=0.0;
        float alpha2=0.0;
        float  central_term=0.0;
        float coef_filt=0;
        float arg=0.0;
        float  coef_wind=0.0;
        int msign,ix;

        float alpha = .54;
        float beta = 6.;
        h_beta = 0.5*beta;
        alpha1=2.*alpha-1.0;
        alpha2=2.*(1.0-alpha);
        central_term=0.0;

        msign=-1;

        for (ix=1; ix <= order/2; ix++){
                msign=-msign ;
                coef_filt = (2.*msign)/(ix*ix);
                arg = PI*ix/(2.*(order/2+2));
                coef_wind=pow((alpha1+alpha2*cos(arg)*cos(arg)),h_beta);
                coef[order/2+ix] = coef_filt*coef_wind;
                central_term = central_term + coef[order/2+ix];
                coef[order/2-ix] = coef[order/2+ix];
        }

        coef[order/2]  = -2.*central_term;

        return;
}

void fd_init_cuda(int order, int nxe, int nze)
{
        mtxBufferLength = (nxe*nze)*sizeof(float);
        coefsBufferLength = (order+1)*sizeof(float);

        // Create a Device pointers
        hipMalloc(&d_p, mtxBufferLength);
        hipMalloc(&d_laplace, mtxBufferLength);
        hipMalloc(&d_coefs_x, coefsBufferLength);
        hipMalloc(&d_coefs_z, coefsBufferLength);

        int div_x, div_z;
        // Set a Grid for the execution on the device
        int tx = ((nxe - 1) / 32 + 1) * 32;
        int tz = ((nze - 1) / 32 + 1) * 32;

        div_x = (float) tx/(float) sizeblock;
        div_z = (float) tz/(float) sizeblock;

        gridx = (int) ceil(div_x);
        gridz = (int) ceil(div_z);
}

void fd_init(int order, int nx, int nz, float dx, float dz)
{
        int io;
        dx2inv = (1./dx)*(1./dx);
        dz2inv = (1./dz)*(1./dz);

        coefs = calc_coefs(order);

        coefs_z = calc_coefs(order);
        coefs_x = calc_coefs(order);

        // pre calc coefs 8 d2 inv
        for (io = 0; io <= order; io++)
        {
                coefs_z[io] = dz2inv * coefs[io];
                coefs_x[io] = dx2inv * coefs[io];
        }

        fd_init_cuda(order,nx,nz);

        return;
}

int main (int argc, char **argv)
{
        read_input(argv[1]);

        printf("Local do arquivo: %s\n", file_path);
        printf("nzb = %i\n", nzb);
        printf("nzb = %i\n", nxb);
        printf("nz = %i\n", nz);
        printf("nx = %i\n", nx);
        printf("dz = %f\n", dz);
        printf("dx = %f\n", dx);
        printf("order = %i\n", order);

        nxe = nx + 2 * nxb;
        nze = nz + 2 * nzb;
        // initialization
        fd_init(order,nxe,nze,dx,dz);

        dim3 dimGrid(gridx, gridz);
        dim3 dimBlock(sizeblock, sizeblock);
        FILE *finput;
        float *input_data;

        if((finput = fopen(file_path, "rb")) == NULL)
                printf("Unable to open file!\n");
        else
                printf("Input successfully opened for reading.\n");

        input_data = (float*)malloc(mtxBufferLength*sizeof(float));
        if(!input_data)
                printf("Input memory allocation error!\n");
        else
                printf("Input memory allocation was successful.\n");

        memset(input_data, 0, mtxBufferLength);

        if( fread(input_data, sizeof(float), nze*nxe, finput) != nze*nxe)
                printf("Input reading error!\n");

        else
                printf("Input reading was successful.\n");
        fclose(finput);

        // data copy
        hipMemcpy(d_p, input_data, mtxBufferLength, hipMemcpyHostToDevice);
        hipMemcpy(d_coefs_x, coefs_x, coefsBufferLength, hipMemcpyHostToDevice);
        hipMemcpy(d_coefs_z, coefs_z, coefsBufferLength, hipMemcpyHostToDevice);

				// kernel utilization
        kernel_lap<<<dimGrid, dimBlock>>>(order,nxe,nze,d_p,d_laplace,d_coefs_x,d_coefs_z);

        float *output_data;
        output_data = (float*)malloc(mtxBufferLength*sizeof(float));
        if(!output_data)
                printf("Output memory allocation error!\n");
        else
                printf("Output memory allocation was successful.\n");
        memset(output_data, 0, mtxBufferLength);
        hipMemcpy(output_data, d_laplace, mtxBufferLength, hipMemcpyDeviceToHost);

        // Writing output
        FILE *foutput;
        if((foutput = fopen("output_cuda.bin", "wb")) == NULL)
                printf("Unable to open file!\n");
        else
                printf("Output successfully opened for writing.\n");

        if( fwrite(output_data, sizeof(float), nze*nxe, foutput) != nze*nxe)
                printf("Output writing error!\n");

        else
                printf("Output writing was successful.\n");
        fclose(foutput);

        // free memory device
				free(file_path);
        free(input_data);
        free(output_data);
        hipFree(d_p);
        hipFree(d_laplace);
        hipFree(d_coefs_x);
        hipFree(d_coefs_z);
        return 0;
}
